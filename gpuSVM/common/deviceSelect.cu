#include "deviceSelect.h"

void chooseLargestGPU(bool verbose) {
  int cudaDeviceCount;
  hipGetDeviceCount(&cudaDeviceCount);
  int cudaDevice = 0;
  int maxSps = 0;
  struct hipDeviceProp_t dp;
  for (int i = 0; i < cudaDeviceCount; i++) {
    hipGetDeviceProperties(&dp, i);
    if (dp.multiProcessorCount > maxSps) {
      maxSps = dp.multiProcessorCount;
      cudaDevice = i;
    }
  }
  hipGetDeviceProperties(&dp, cudaDevice);
  if (verbose) {
    printf("Using cuda device %i: %s\n", cudaDevice, dp.name);
  }
  hipSetDevice(cudaDevice);
}
