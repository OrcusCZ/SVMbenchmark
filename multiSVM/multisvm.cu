#define Cache multisvmCache

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <cutil.h>
//#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include "src/common/cuTimer.cu"
#include "src/common/parseinputs.cpp"
#include "src/training/training.cu"
//#include "src/testing/testing.cu"

