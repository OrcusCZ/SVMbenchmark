#include "hip/hip_runtime.h"

#ifndef _TRAINING_KERNEL_H_
#define _TRAINING_KERNEL_H_

#include <stdio.h>

/**
 * Set initial values of the binary labels and alphas
 * @param d_ltraindata device pointer to multiclass labels
 * @param d_rdata device pointer to the binary matrix that encodes the output code
 * @param d_ytraindata device pointer to the array with binary labels
 * @param d_atraindata device pointer to the array with the alphas
 * @param d_fdata device pointer to the intermediate values of f
 * @param ntraining number of training samples in the training set
 * @param ntasks number of binary tasks to be solved
 * @param d_active device pointer to the binary array that indicates the status of the task
 */
template <unsigned int blockSize, bool isNtrainingPow2>
__global__ static void initializetraining(	int* d_ltraindata,
											int* d_rdata,
											int* d_ytraindata,
											float* d_atraindata,
											float* d_fdata,
											int ntraining,
											int ntasks,
											int* d_active)
{
	const unsigned int j = blockIdx.y;
	unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
	const unsigned int gridSize = blockSize*2*gridDim.x;

	const unsigned int bidy= d_active[j];

	while (i < ntraining)
	{
		int label= d_ltraindata[i];
		d_ytraindata[bidy*ntraining + i]= d_rdata[(label-1)*ntasks + bidy];
		d_atraindata[bidy* ntraining + i]=0.0f;
		d_fdata[bidy* ntraining + i]= -1.0* (float)( d_ytraindata[bidy*ntraining + i]);

		if (isNtrainingPow2 || i + blockSize < ntraining)
		{
			label= d_ltraindata[i + blockSize];
			d_ytraindata[bidy*ntraining + i + blockSize]= d_rdata[(label-1)*ntasks + bidy];
			d_atraindata[bidy* ntraining + i + blockSize]=0.0f;
			d_fdata[bidy* ntraining + i + blockSize]= -1.0* (float)( d_ytraindata[bidy*ntraining + i + blockSize]);

		}
		i += gridSize;
	}
	__syncthreads();

}

/**
 * Calculate the new values of the chosen duple of alphas
 * @param d_xtraindata device pointer to the training set
 * @param d_kdata device pointer to the cached rows of the gram matrix
 * @param d_ytraindata device pointer to the array with binary labels
 * @param d_atraindata device pointer to the array with the alphas
 * @param d_anewtraindata device pointer to the new duple of alphas (alpha 1 and alpha 2)
 * @param d_aoldtraindata device pointer to the old duple of alphas (alpha 1 and alpha 2)
 * @param d_fdata device pointer to the intermediate values of f
 * @param d_Iup_global device pointer to the Iup indexes for each binary task
 * @param d_Ilow_global device pointer to the Ilow indexes for each binary task
 * @param d_Iup_cache device pointer to the location of the Iup index in the kernel cache
 * @param d_Ilow_cache device pointer to the location of the Ilow index in the kernel cache
 * @param d_done device pointer containing the status of each binary task
 * @param ntraining number of training samples in the training set
 * @param nfeatures number of features in each of the training samples
 * @param ntasks number of binary tasks to be solved
 * @param d_C device pointer to the regularization parameter for each binary task
 */
__global__ static void calculatealphas(		float* d_xtraindata,
											float* d_kdata,
											int* d_ytraindata,
											float* d_atraindata,
											float* d_anewtraindata,
											float* d_aoldtraindata,
											float* d_fdata,
											int* d_Iup_global,
											int* d_Ilow_global,
											int* d_Iup_cache,
											int* d_Ilow_cache,
											int* d_done,
											int ntraining,
											int nfeatures,
											int ntasks,
											float* d_C)
{

	const unsigned int tid = threadIdx.x;
	const unsigned int bidy = blockIdx.x;
	const float eps= 0.000001;

	//Check if the task has converged
	if(d_done[tid]==0)
	{

		int blockYAlpha=(int) ceil((float)(ntasks)/(float)(TPB));

		if((bidy != blockYAlpha-1) || ((bidy == blockYAlpha-1) && tid < (ntasks - bidy*TPB)))
		{
			//Calculate alpha_2 and alpha _1

			float C= d_C[tid];

			int g_Iup=d_Iup_global[tid];
			int g_Ilow=d_Ilow_global[tid];

			int y_2= d_ytraindata[tid*ntraining + g_Iup];
			int y_1= d_ytraindata[tid*ntraining + g_Ilow];

			float alpha_2_old= d_atraindata[tid*ntraining + g_Iup];
			float alpha_1_old= d_atraindata[tid*ntraining + g_Ilow];

			d_aoldtraindata[tid*2]= alpha_1_old;
			d_aoldtraindata[tid*2 +1]= alpha_2_old;


			float f_2_old= d_fdata[tid*ntraining + g_Iup];
			float f_1_old= d_fdata[tid*ntraining + g_Ilow];

			int s= y_1 * y_2;


			float gamma=0.0f;
			float L=0.0f;
			float H=0.0f;

			if(y_1 == y_2)
			{
				gamma= alpha_1_old + alpha_2_old;
			}
			else
			{
				gamma= alpha_1_old - alpha_2_old;
			}

			if(s==1)
			{
				L= max( 0.0f, gamma -C);
				H= min (C,gamma);
			}
			else
			{
				L= max(0.0f, -gamma);
				H= min(C, C -gamma);
			}


			if(H<=L)
			{
				d_done[tid]=1;

			}


			float K12= d_kdata [d_Ilow_cache[tid]*ntraining + g_Iup];
			float K11= d_kdata [d_Ilow_cache[tid]*ntraining + g_Ilow];
			float K22= d_kdata [d_Iup_cache[tid]*ntraining + g_Iup];

			float nu= 2*K12 - K11 -K22;


			float alpha_2_new=0.0f;
			float alpha_1_new=0.0f;

			if(nu < 0)
			{
				alpha_2_new= alpha_2_old - (y_2*(f_1_old - f_2_old)/nu);
				if(alpha_2_new <L)
				{
					alpha_2_new=L;
				}
				else if (alpha_2_new>H)
				{
					alpha_2_new=H;
				}
			}
			else
			{
				float slope= y_2 *(f_1_old - f_2_old);
				float change= slope * (H-L);
				if(fabs(change)>0.0f)
				{
					if(slope>0.0f)
					{
						alpha_2_new= H;
					}
					else
					{
						alpha_2_new= L;
					}
				}
				else
				{
					alpha_2_new= alpha_2_old;
				}

				if( alpha_2_new > C - eps * C)
				{
					alpha_2_new=C;
				}
				else if (alpha_2_new < eps * C)
				{
					alpha_2_new=0.0f;
				}
			}

			if( fabs( alpha_2_new - alpha_2_old) < eps * ( alpha_2_new + alpha_2_old + eps))
			{
				d_done[tid]=1;
			}

			if(s==1)
			{
				alpha_1_new= gamma - alpha_2_new;
			}
			else
			{
				alpha_1_new= gamma + alpha_2_new;
			}

			if( alpha_1_new > C - eps * C)
			{
				alpha_1_new=C;
			}
			else if (alpha_1_new < eps * C)
			{
				alpha_1_new=0.0f;
			}

			d_anewtraindata[tid*2]= alpha_1_new;
			d_anewtraindata[tid*2 +1]= alpha_2_new;
			//printf("a1 = %.3f a2 = %.3f\n", alpha_1_new, alpha_2_new); //LLLLLLLLLLLLLL

		}

	}
	__syncthreads();
}

/**
 * Calculate the new values of the chosen duple of alphas
 * @param d_xtraindata device pointer to the training set
 * @param d_kdata device pointer to the cached rows of the gram matrix
 * @param d_ytraindata device pointer to the array with binary labels
 * @param d_atraindata device pointer to the array with the alphas
 * @param d_anewtraindata device pointer to the new duple of alphas (alpha 1 and alpha 2)
 * @param d_aoldtraindata device pointer to the old duple of alphas (alpha 1 and alpha 2)
 * @param d_fdata device pointer to the intermediate values of f
 * @param d_Iup_global device pointer to the Iup indexes for each binary task
 * @param d_Ilow_global device pointer to the Ilow indexes for each binary task
 * @param d_Iup_cache device pointer to the location of the Iup index in the kernel cache
 * @param d_Ilow_cache device pointer to the location of the Ilow index in the kernel cache
 * @param d_done device pointer containing the status of each binary task
 * @param ntraining number of training samples in the training set
 * @param nfeatures number of features in each of the training samples
 * @param activeTasks number of non converged tasks
 * @param ntasks number of binary tasks to be solved
 * @param d_C device pointer to the regularization parameter for each binary task
 */
template <unsigned int blockSize, bool isNtrainingPow2>
__global__ static void updateparams(		float* d_xtraindata,
											float* d_kdata,
											int* d_ytraindata,
											float* d_atraindata,
											float* d_anewtraindata,
											float* d_aoldtraindata,
											float* d_fdata,
											int* d_Iup_global,
											int* d_Ilow_global,
											int* d_Iup_cache,
											int* d_Ilow_cache,
											int* d_done,
											int* d_active,
											int ntraining,
											int nfeatures,
											int ntasks,
											int activeTasks,
											float* d_C)
{

	const unsigned int tid = threadIdx.x;
	const unsigned int bidx = blockIdx.x;
	unsigned int j = blockIdx.y;
	unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;


	int bidy= d_active[j];

	if(d_done[bidy]==0)
	{
		int g_Iup=d_Iup_global[bidy];
		int g_Ilow=d_Ilow_global[bidy];

		float alpha_1_new= d_anewtraindata[bidy*2];
		float alpha_2_new= d_anewtraindata[bidy*2+1];

		float alpha_1_old= d_aoldtraindata[bidy*2];
		float alpha_2_old= d_aoldtraindata[bidy*2 +1];

		int y_2= d_ytraindata[bidy*ntraining + g_Iup];
		int y_1= d_ytraindata[bidy*ntraining + g_Ilow];

		while (i < ntraining)
		{

			float f_i_old= d_fdata[bidy* ntraining +i];

			float K1i= d_kdata [d_Ilow_cache [bidy] * ntraining +i];
			float K2i= d_kdata [d_Iup_cache [bidy] * ntraining + i];

			float f_i_new= f_i_old + (alpha_1_new - alpha_1_old)*y_1* K1i + (alpha_2_new - alpha_2_old)*y_2* K2i;

			d_fdata[bidy*ntraining +  i]= f_i_new;


			if (isNtrainingPow2 || i + blockSize < ntraining)
			{
				f_i_old= d_fdata[bidy* ntraining +i + blockSize];
				K1i= d_kdata [d_Ilow_cache [bidy] * ntraining + i + blockSize];
				K2i= d_kdata [d_Iup_cache [bidy] * ntraining + i + blockSize];

				f_i_new= f_i_old + (alpha_1_new - alpha_1_old)*y_1* K1i + (alpha_2_new - alpha_2_old)*y_2* K2i;

				d_fdata[bidy*ntraining +  i + blockSize]= f_i_new;
			}

			 i += gridSize;
		}

		if(bidx==0 && tid==0)
		{
			d_atraindata[bidy*ntraining + g_Ilow]= d_anewtraindata[bidy*2];
			d_atraindata[bidy*ntraining + g_Iup]= d_anewtraindata[bidy*2+1];
		}


	}
}


#endif // _TRAINING_KERNEL_H_
