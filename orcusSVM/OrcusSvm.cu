#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "OrcusSvm.h"
#include "cudaerror.cuh"
#include "debug.h"

template<typename T>
T getgriddim(T totallen, T blockdim)
{
    return (totallen + blockdim - (T)1) / blockdim;
}

template<typename T>
T rounduptomult(T x, T m)
{
    return ((x + m - (T)1) / m) * m;
}

__global__ void kernelInitg(float * g, int len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len)
        g[i] = 1;
}

__global__ void kernelComputeK(float * K, const float * x, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x,
        j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < num_vec && j < num_vec)
    {
        float sum = 0;
        for (int d = 0; d < dim; d++)
        {
            float diff = x[dim_aligned * i + d] - x[dim_aligned * j + d];
            sum += diff * diff;
        }
        K[num_vec_aligned * j + i] = exp(-gamma * sum);
    }
#ifdef _DEBUG
    else
        K[num_vec_aligned * j + i] = 0;
#endif
}

__global__ void kernelSelectI(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < num_vec)
    {
        float y_ = y[i];
        float a_ = alpha[i];
        if ((y_ == 1 && a_ < C) || (y_ == -1 && a_ > 0))
            valbuf[i] = y[i] * g[i];
        else
            valbuf[i] = -FLT_MAX;
        idxbuf[i] = i;
    }
    else
        valbuf[i] = -FLT_MAX;
}

//first order search
__global__ void kernelSelectJ1(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < num_vec)
    {
        float y_ = y[i];
        float a_ = alpha[i];
        if ((y_ == 1 && a_ > 0) || (y_ == -1 && a_ < C))
            valbuf[i] = -y[i] * g[i]; //return negative, so we can use reducemax
        else
            valbuf[i] = -FLT_MAX;
        idxbuf[i] = i;
    }
    else
        valbuf[i] = -FLT_MAX;
}

__global__ void kernelReduceMaxIdx(float * val, int * idx, float * val_out, int * idx_out, int len)
{
    extern __shared__ float sval[];
    int * sidx = (int *)(sval + blockDim.x);

    int frame = blockDim.x * blockIdx.x,
        iter = 0;
    while (frame < len)
    {
        int i = frame + threadIdx.x;
        if (i < len)
        {
            sval[threadIdx.x] = val[i];
            sidx[threadIdx.x] = idx[i];
        }
        else
        {
            sval[threadIdx.x] = -FLT_MAX;
        }
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (threadIdx.x < s)
            {
                if (sval[threadIdx.x + s] > sval[threadIdx.x])
                {
                    sval[threadIdx.x] = sval[threadIdx.x + s];
                    sidx[threadIdx.x] = sidx[threadIdx.x + s];
                }
            }
            __syncthreads();
        }

        if (threadIdx.x == 0)
        {
            int shift = iter * gridDim.x;
            val_out[shift + blockIdx.x] = sval[0];
            idx_out[shift + blockIdx.x] = sidx[0];
        }
        __syncthreads();
        frame += gridDim.x * blockDim.x;
        iter++;
    }
}

void reduceMaxIdx(float * d_val, int * d_idx, float * d_val2, int * d_idx2, int len, int reduce_block_size)
{
    //int orig_len = len;
    /*dim3 dimBlock = dim3(reduce_block_size);
    while (len > 1)
    {
        dim3 dimGrid = dim3(getgriddim(len, (int)dimBlock.x));
        kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val, d_idx, d_val2, d_idx2, len);
        len = dimGrid.x;
    }*/
    dim3 dimBlock = dim3(reduce_block_size);
    dim3 dimGrid = dim3(std::min(256, getgriddim(len, (int)dimBlock.x)));
    kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val, d_idx, d_val2, d_idx2, len);
    kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val2, d_idx2, d_val, d_idx, dimGrid.x);
    //export_cuda_buffer(d_val, 1, orig_len, sizeof(float), "reduceval.dat");
    //export_cuda_buffer(d_idx, 1, orig_len, sizeof(int), "reduceidx.dat");
}

__global__ void kernelComputeLambda(float * lambda, const float * y, const float * g, const float * K, const float * alpha, float C, const int * ws, int num_vec_aligned)
{
    int i = ws[0];
    int j = ws[1];
    float l1 = y[i] == 1 ? C - alpha[i] : alpha[i];
    float l2 = y[j] == 1 ? alpha[j] : C - alpha[j];
    float l3 = (y[i] * g[i] - y[j] * g[j]) / (K[num_vec_aligned * i + i] + K[num_vec_aligned * j + j] - 2 * K[num_vec_aligned * i + j]);
    *lambda = min(l1, min(l2, l3));
}

__global__ void kernelUpdateg(float * g, const float * lambda, const float * y, const float * K, const int * ws, int num_vec, int num_vec_aligned)
{
    int i = ws[0];
    int j = ws[1];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        g[k] += *lambda * y[k] * (K[num_vec_aligned * j + k] - K[num_vec_aligned * i + k]);
    }
}

__global__ void kernelUpdateAlpha(float * alpha, const float * lambda, const float * y, const int * ws, int num_vec)
{
    int i = ws[0];
    int j = ws[1];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        float l = *lambda;
        alpha[i] += l * y[i];
        alpha[j] -= l * y[j];
    }
}

void OrcusSvmTrain(float * alpha, float * rho, const float * x, const float * y, size_t num_vec, size_t num_vec_aligned, size_t dim, size_t dim_aligned, float C, float gamma, float eps)
{
    float *d_alpha = nullptr,
        *d_x = nullptr,
        *d_y = nullptr,
        *d_g = nullptr,
        *d_K = nullptr,
        *d_reduceval = nullptr,
        *d_reduceval2 = nullptr;
    int *d_reduceidx = nullptr,
        *d_reduceidx2 = nullptr;
    float *d_lambda = nullptr;
    int *d_workingset = nullptr;

    size_t reduce_block_size = 256;
    size_t reduce_buff_size = rounduptomult(num_vec, reduce_block_size);

    assert_cuda(hipMalloc(&d_alpha, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_x, num_vec_aligned * dim_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_y, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_g, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_K, num_vec_aligned * num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceval, reduce_buff_size * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceidx, reduce_buff_size * sizeof(int)));
    assert_cuda(hipMalloc(&d_reduceval2, reduce_buff_size / reduce_block_size * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceidx2, reduce_buff_size / reduce_block_size * sizeof(int)));
    assert_cuda(hipMalloc(&d_lambda, sizeof(float)));
    assert_cuda(hipMalloc(&d_workingset, 2 * sizeof(int)));

    assert_cuda(hipMemset(d_alpha, 0, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMemcpy(d_x, x, num_vec_aligned * dim_aligned * sizeof(float), hipMemcpyHostToDevice));
    assert_cuda(hipMemcpy(d_y, y, num_vec_aligned * sizeof(float), hipMemcpyHostToDevice));

    export_cuda_buffer(d_x, dim_aligned, num_vec_aligned, sizeof(float), "x.dat");

    dim3 dimBlock(256);
    dim3 dimGrid(getgriddim(num_vec_aligned, (size_t)dimBlock.x));
    kernelInitg<<<dimGrid, dimBlock>>>(d_g, num_vec_aligned);

    export_cuda_buffer(d_g, num_vec_aligned, 1, sizeof(float), "g.dat");

    dimBlock = dim3(16, 16);
    dimGrid = dim3(getgriddim(num_vec_aligned, (size_t)dimBlock.x), getgriddim(num_vec_aligned, (size_t)dimBlock.y));
    kernelComputeK<<<dimGrid, dimBlock>>>(d_K, d_x, gamma, num_vec, num_vec_aligned, dim, dim_aligned);

    export_cuda_buffer(d_K, num_vec_aligned, num_vec_aligned, sizeof(float), "K.dat");

    dimBlock = dim3(reduce_block_size);
    dimGrid = dim3(getgriddim(num_vec_aligned, (size_t)dimBlock.x));
    for (int iter = 0;; iter++)
    {
        kernelSelectI<<<dimGrid, dimBlock>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec);
        //export_cuda_buffer(d_reduceval, 1, reduce_buff_size, sizeof(float), "reduceval.dat");
        //export_cuda_buffer(d_reduceidx, 1, reduce_buff_size, sizeof(int), "reduceidx.dat");
        reduceMaxIdx(d_reduceval, d_reduceidx, d_reduceval2, d_reduceidx2, num_vec_aligned, reduce_block_size);
        assert_cuda(hipMemcpy(d_workingset, d_reduceidx, sizeof(int), hipMemcpyDeviceToDevice));

        kernelSelectJ1<<<dimGrid, dimBlock>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec);
        //export_cuda_buffer(d_reduceval, 1, reduce_buff_size, sizeof(float), "reduceval.dat");
        //export_cuda_buffer(d_reduceidx, 1, reduce_buff_size, sizeof(int), "reduceidx.dat");
        reduceMaxIdx(d_reduceval, d_reduceidx, d_reduceval2, d_reduceidx2, num_vec_aligned, reduce_block_size);
        assert_cuda(hipMemcpy(d_workingset + 1, d_reduceidx, sizeof(int), hipMemcpyDeviceToDevice));

        if (iter % 1000 == 0)
        {
            int ws[2];
            float yi, yj, gi, gj;
            assert_cuda(hipMemcpy(&ws, d_workingset, 2 * sizeof(int), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&yi, d_y + ws[0], sizeof(float), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&yj, d_y + ws[1], sizeof(float), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&gi, d_g + ws[0], sizeof(float), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&gj, d_g + ws[1], sizeof(float), hipMemcpyDeviceToHost));
            float diff = yi * gi - yj * gj;
            std::cout << "Iter " << iter << ": " << diff << std::endl;
            if (diff < eps)
            {
                *rho = (yi * gi + yj * gj) / 2;
                std::cout << "Optimality reached, stopping loop. rho = " << *rho << std::endl;
                break;
            }
        }

        kernelComputeLambda<<<1, 1>>>(d_lambda, d_y, d_g, d_K, d_alpha, C, d_workingset, num_vec_aligned);
        kernelUpdateg<<<dimGrid, dimBlock>>>(d_g, d_lambda, d_y, d_K, d_workingset, num_vec, num_vec_aligned);
        kernelUpdateAlpha<<<1, 1>>>(d_alpha, d_lambda, d_y, d_workingset, num_vec);

        //float lambda;
        //int ws[2];
        //assert_cuda(hipMemcpy(&lambda, d_lambda, sizeof(float), hipMemcpyDeviceToHost));
        //assert_cuda(hipMemcpy(&ws, d_workingset, 2 * sizeof(int), hipMemcpyDeviceToHost));
        //std::cout << "i: " << ws[0] << ", j: " << ws[1] << ", lambda: " << lambda << std::endl;
    }

    assert_cuda(hipMemcpy(alpha, d_alpha, num_vec * sizeof(float), hipMemcpyDeviceToHost));

    assert_cuda(hipFree(d_alpha));
    assert_cuda(hipFree(d_x));
    assert_cuda(hipFree(d_y));
    assert_cuda(hipFree(d_g));
    assert_cuda(hipFree(d_K));
    assert_cuda(hipFree(d_reduceval));
    assert_cuda(hipFree(d_reduceidx));
    assert_cuda(hipFree(d_reduceval2));
    assert_cuda(hipFree(d_reduceidx2));
    assert_cuda(hipFree(d_lambda));
}