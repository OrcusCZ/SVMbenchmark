#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "OrcusSvm.h"
#include "cudaerror.cuh"
#include "debug.h"

template<typename T>
T getgriddim(T totallen, T blockdim)
{
    return (totallen + blockdim - (T)1) / blockdim;
}

template<typename T>
T rounduptomult(T x, T m)
{
    return ((x + m - (T)1) / m) * m;
}

__global__ void dummyKernel()
{
}

template<typename T>
__global__ void kernelMemset(T * mem, T v, int n)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    while (k < n)
    {
        mem[k] = v;
        k += gridDim.x * blockDim.x;
    }
}

template<typename T>
void memsetCuda(T * d_mem, T v, int n)
{
    dim3 dimBlock(256);
    dim3 dimGrid(std::min(2048, getgriddim<int>(n, dimBlock.x)));
    kernelMemset<T><<<dimGrid, dimBlock>>>(d_mem, v, n);
}

__global__ void kernelComputeK(float * K, const float * x, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x,
        j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < num_vec && j < num_vec)
    {
        float sum = 0;
        for (int d = 0; d < dim; d++)
        {
            float diff = x[dim_aligned * i + d] - x[dim_aligned * j + d];
            sum += diff * diff;
        }
        K[num_vec_aligned * j + i] = exp(-gamma * sum);
    }
#ifdef _DEBUG
    else
        K[num_vec_aligned * j + i] = 0;
#endif
}

__global__ void kernelComputeKv2(float * K, const float * x, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x,
        j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < num_vec)
    {
        while (j < num_vec)
        {
            float sum = 0;
            for (int d = 0; d < dim; d++)
            {
                float diff = x[dim_aligned * i + d] - x[dim_aligned * j + d];
                sum += diff * diff;
            }
            K[num_vec_aligned * j + i] = exp(-gamma * sum);
            j += gridDim.y * blockDim.y;
        }
    }
}

__global__ void kernelPow2(float * x2, const float * x, int w, int h, int pitch)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x,
        j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < w && j < h)
    {
        int idx = pitch * j + i;
        float v = x[idx];
        x2[idx] = v * v;
    }
}

__global__ void kernelRBFExp(float * K, float gamma, int num_vec, int pitch)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x,
        j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < num_vec && j < num_vec)
    {
        int idx = pitch * j + i;
        K[idx] = exp(-gamma * K[idx]);
    }
}

void computeK(float * d_K, const float * d_x, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned, hipblasHandle_t cublas)
{
    //dim3 dimBlock(16, 16);
    //dim3 dimGrid(getgriddim(num_vec_aligned, (int)dimBlock.x));
    //kernelComputeKv2<<<dimGrid, dimBlock>>>(d_K, d_x, gamma, /*num_vec*/1000, num_vec_aligned, dim, dim_aligned);

    float *d_x2 = nullptr,
        *d_x2sum = nullptr,
        *d_ones = nullptr;

    int ones_size = std::max(num_vec_aligned, dim_aligned);
    assert_cuda(hipMalloc(&d_x2, num_vec_aligned * dim_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_x2sum, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_ones, ones_size * sizeof(float)));

    //float one_val = 1;
    //assert_cuda(hipMemset(d_ones, *(int *)&one_val, ones_size * sizeof(float)));
    //hipMemsetD32(d_ones, *(int *)&one_val, ones_size);
    memsetCuda<float>(d_ones, 1, ones_size);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(getgriddim(dim, (int)dimBlock.x), getgriddim(num_vec, (int)dimBlock.y));
    kernelPow2<<<dimGrid, dimBlock>>>(d_x2, d_x, dim, num_vec, dim_aligned);
    float a = 1,
        b = 0;
    assert_cublas(hipblasSgemv(cublas, HIPBLAS_OP_T, dim, num_vec, &a, d_x2, dim_aligned, d_ones, 1, &b, d_x2sum, 1));
    a = -2;
    assert_cublas(hipblasSgemm(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, num_vec, num_vec, dim, &a, d_x, dim_aligned, d_x, dim_aligned, &b, d_K, num_vec_aligned));

    a = 1;
    assert_cublas(hipblasSger(cublas, num_vec, num_vec, &a, d_x2sum, 1, d_ones, 1, d_K, num_vec_aligned));
    assert_cublas(hipblasSger(cublas, num_vec, num_vec, &a, d_ones, 1, d_x2sum, 1, d_K, num_vec_aligned));

    dimGrid = dim3(getgriddim(num_vec, (int)dimBlock.x), getgriddim(num_vec, (int)dimBlock.y));
    kernelRBFExp<<<dimGrid, dimBlock>>>(d_K, gamma, num_vec, num_vec_aligned);

    assert_cuda(hipFree(d_x2));
    assert_cuda(hipFree(d_x2sum));
    assert_cuda(hipFree(d_ones));
}

void computeKDiag(float * d_KDiag, int num_vec)
{
    //K[i,i] is always 1 for RBF kernel, let's just use memset here
    memsetCuda<float>(d_KDiag, 1, num_vec);
}

__global__ void kernelSelectI(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    if (k < num_vec)
    {
        float y_ = y[k];
        float a_ = alpha[k];
        if ((y_ == 1 && a_ < C) || (y_ == -1 && a_ > 0))
            valbuf[k] = y[k] * g[k];
        else
            valbuf[k] = -FLT_MAX;
        idxbuf[k] = k;
    }
    else
        valbuf[k] = -FLT_MAX;
}

//first order search
__global__ void kernelSelectJ1(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    if (k < num_vec)
    {
        float y_ = y[k];
        float a_ = alpha[k];
        if ((y_ == 1 && a_ > 0) || (y_ == -1 && a_ < C))
            valbuf[k] = -y[k] * g[k]; //return negative, so we can use reducemax
        else
            valbuf[k] = -FLT_MAX;
        idxbuf[k] = k;
    }
    else
        valbuf[k] = -FLT_MAX;
}

//second order search
__global__ void kernelSelectJ(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec, int num_vec_aligned, const int * i_ptr, const float * K)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    if (k < num_vec)
    {
        int i = *i_ptr;
        float y_ = y[k];
        float a_ = alpha[k];
        float th = y[i] * g[i];
        if (((y_ == 1 && a_ > 0) || (y_ == -1 && a_ < C)) && th > y[k] * g[k])
        {
            float den = K[num_vec_aligned * i + i] + K[num_vec_aligned * k + k] - 2 * K[num_vec_aligned * i + k];
            float v = th - y[k] * g[k];
            valbuf[k] = v * v / den;
        }
        else
            valbuf[k] = -FLT_MAX;
        idxbuf[k] = k;
    }
    else
        valbuf[k] = -FLT_MAX;
}

//second order search with cached K
__global__ void kernelSelectJCached(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec, int num_vec_aligned, const int * i_ptr, const float * K, const float * KDiag, const int * KCacheRemapIdx)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    if (k < num_vec)
    {
        int i = *i_ptr;
        int cache_row = KCacheRemapIdx[i];
        float y_ = y[k];
        float a_ = alpha[k];
        float th = y[i] * g[i];
        if (((y_ == 1 && a_ > 0) || (y_ == -1 && a_ < C)) && th > y[k] * g[k])
        {
            float den = KDiag[i] + KDiag[k] - 2 * K[num_vec_aligned * cache_row + k];
            float v = th - y[k] * g[k];
            valbuf[k] = v * v / den;
        }
        else
            valbuf[k] = -FLT_MAX;
        idxbuf[k] = k;
    }
    else
        valbuf[k] = -FLT_MAX;
}

__global__ void kernelReduceMaxIdx(float * val, int * idx, float * val_out, int * idx_out, int len)
{
    extern __shared__ float sval[];
    int * sidx = (int *)(sval + blockDim.x);

    int frame = blockDim.x * blockIdx.x,
        iter = 0;
    while (frame < len)
    {
        int i = frame + threadIdx.x;
        if (i < len)
        {
            sval[threadIdx.x] = val[i];
            sidx[threadIdx.x] = idx[i];
        }
        else
        {
            sval[threadIdx.x] = -FLT_MAX;
        }
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (threadIdx.x < s)
            {
                if (sval[threadIdx.x + s] > sval[threadIdx.x])
                {
                    sval[threadIdx.x] = sval[threadIdx.x + s];
                    sidx[threadIdx.x] = sidx[threadIdx.x + s];
                }
            }
            __syncthreads();
        }

        if (threadIdx.x == 0)
        {
            int shift = iter * gridDim.x;
            val_out[shift + blockIdx.x] = sval[0];
            idx_out[shift + blockIdx.x] = sidx[0];
        }
        __syncthreads();
        frame += gridDim.x * blockDim.x;
        iter++;
    }
}

void reduceMaxIdx(float * d_val, int * d_idx, float * d_val2, int * d_idx2, int len, int reduce_block_size)
{
    //int orig_len = len;
    /*dim3 dimBlock = dim3(reduce_block_size);
    while (len > 1)
    {
        dim3 dimGrid = dim3(getgriddim(len, (int)dimBlock.x));
        kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val, d_idx, d_val2, d_idx2, len);
        len = dimGrid.x;
    }*/
    dim3 dimBlock = dim3(reduce_block_size);
    dim3 dimGrid = dim3(std::min(256, getgriddim(len, (int)dimBlock.x)));
    //dummyKernel<<<dimGrid, dimBlock>>>();
    kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val, d_idx, d_val2, d_idx2, len);
    len = dimGrid.x;
    dimGrid.x = std::min(reduce_block_size, (int)getgriddim(dimGrid.x, dimBlock.x));
    kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val2, d_idx2, d_val, d_idx, len);
    //export_cuda_buffer(d_val, 1, orig_len, sizeof(float), "reduceval.dat");
    //export_cuda_buffer(d_idx, 1, orig_len, sizeof(int), "reduceidx.dat");
}

__global__ void kernelUpdateg(float * g, const float * lambda, const float * y, const float * K, const int * ws, int num_vec, int num_vec_aligned)
{
    int i = ws[0];
    int j = ws[1];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        g[k] += *lambda * y[k] * (K[num_vec_aligned * j + k] - K[num_vec_aligned * i + k]);
    }
}

__global__ void kernelUpdategCached(float * g, const float * lambda, const float * y, const float * K, const int * ws, int num_vec, int num_vec_aligned, const int * KCacheRemapIdx)
{
    int i = ws[0];
    int j = ws[1];
    int i_cache_row = KCacheRemapIdx[i];
    int j_cache_row = KCacheRemapIdx[j];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        g[k] += *lambda * y[k] * (K[num_vec_aligned * j_cache_row + k] - K[num_vec_aligned * i_cache_row + k]);
    }
}

__global__ void kernelUpdateAlphaAndLambda(float * alpha, float * lambda, const float * y, const float * g, const float * K, float C, const int * ws, int num_vec, int num_vec_aligned)
{
    int i = ws[0];
    int j = ws[1];
    float l1 = y[i] > 0 ? C - alpha[i] : alpha[i];
    float l2 = y[j] > 0 ? alpha[j] : C - alpha[j];
    float l3 = (y[i] * g[i] - y[j] * g[j]) / (K[num_vec_aligned * i + i] + K[num_vec_aligned * j + j] - 2 * K[num_vec_aligned * i + j]);
    float l = min(l1, min(l2, l3));

    *lambda = l;
    alpha[i] += l * y[i];
    alpha[j] -= l * y[j];
}

__global__ void kernelUpdateAlphaAndLambdaCached(float * alpha, float * lambda, const float * y, const float * g, const float * K, float C, const int * ws, int num_vec, int num_vec_aligned, const float * KDiag, const int * KCacheRemapIdx)
{
    int i = ws[0];
    int j = ws[1];
    int cache_row = KCacheRemapIdx[i];
    float l1 = y[i] > 0 ? C - alpha[i] : alpha[i];
    float l2 = y[j] > 0 ? alpha[j] : C - alpha[j];
    float l3 = (y[i] * g[i] - y[j] * g[j]) / (KDiag[i] + KDiag[j] - 2 * K[num_vec_aligned * cache_row + j]);
    float l = min(l1, min(l2, l3));

    *lambda = l;
    alpha[i] += l * y[i];
    alpha[j] -= l * y[j];
}

__device__ int d_cacheUpdateCnt;

__global__ void kernelCheckCache(const int * i_ptr, float * K, int * KCacheRemapIdx, int * KCacheRowIdx, int * KCacheRowPriority, int cache_rows, const float * x, const float * xT, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned, int lastPtrIdx)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = *i_ptr;
    if (KCacheRemapIdx[i] >= 0)
    {
        if (j == 0)
            KCacheRowIdx[cache_rows + (1 - lastPtrIdx)] = KCacheRowIdx[cache_rows + lastPtrIdx];
        return;  //item already in cache
    }
    int last = (KCacheRowIdx[cache_rows + lastPtrIdx] + 1) % cache_rows;
    if (j == 0)
    {
        KCacheRowIdx[cache_rows + (1 - lastPtrIdx)] = last;
        int del_i = KCacheRowIdx[last];
        if (del_i >= 0)
            KCacheRemapIdx[del_i] = -1;  //cache row for vector [del_i] will be overwritten, remove it from RemapIdx array
        //set correct indices
        KCacheRemapIdx[i] = last;
        KCacheRowIdx[last] = i;
        d_cacheUpdateCnt++;
    }

    //calculate cache matrix row [last], original index is [i]
    extern __shared__ float sx[];
    for (int idxshift = 0; idxshift < dim; idxshift += blockDim.x)
    {
        int idx = idxshift + threadIdx.x;
        if (idx < dim)
            //xi[idx] = xT[num_vec_aligned * idx + i];
            sx[idx] = x[dim_aligned * i + idx];
    }
    __syncthreads();
    while (j < num_vec)
    {
        float sum = 0;
        for (int d = 0; d < dim; d++)
        {
            //float diff = xi[d] - x[dim_aligned * j + d];
            float diff = sx[d] - xT[num_vec_aligned * d + j];
            //float diff = x[dim_aligned * i + d] - x[dim_aligned * j + d];
            //float diff = xT[num_vec_aligned * d + i] - xT[num_vec_aligned * d + j];
            sum += diff * diff;
        }
        K[num_vec_aligned * last + j] = expf(-gamma * sum);
        j += gridDim.x * blockDim.x;
    }
}

__global__ void kernelCheckCache_(const int * i_ptr, float * K, int * KCacheRemapIdx, int * KCacheRowIdx, int * KCacheRowPriority, int cache_rows, const float * x, const float * xT, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned, int lastPtrIdx)
{
    extern __shared__ int2 spriority[];
    int i = *i_ptr;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if (KCacheRemapIdx[i] >= 0)
    {
        if (j == 0)
            KCacheRowPriority[KCacheRemapIdx[i]] = d_cacheUpdateCnt;  // refresh priority
        return;  //item already in cache
    }
    //int last = (KCacheRowIdx[cache_rows + lastPtrIdx] + 1) % cache_rows;
    int2 minpriority = make_int2(INT_MAX, 0);
    for (int k = 0; k < cache_rows; k += blockDim.x)
    {
        int idx = k + threadIdx.x;
        if (idx < cache_rows)
        {
            int v = KCacheRowPriority[idx];
            if (v < minpriority.x)
                minpriority = make_int2(v, idx);
        }
    }
    spriority[threadIdx.x] = minpriority;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (spriority[threadIdx.x + s].x < spriority[threadIdx.x].x)
                spriority[threadIdx.x] = spriority[threadIdx.x + s];
        }
        __syncthreads();
    }
    int last = spriority[0].y;
    if (j == 0)
    {
        //KCacheRowIdx[cache_rows + (1 - lastPtrIdx)] = last;
        int del_i = KCacheRowIdx[last];
        if (del_i >= 0)
            KCacheRemapIdx[del_i] = -1;  //cache row for vector [del_i] will be overwritten, remove it from RemapIdx array
        //set correct indices
        KCacheRemapIdx[i] = last;
        KCacheRowIdx[last] = i;
        KCacheRowPriority[last] = ++d_cacheUpdateCnt;
    }

    //calculate cache matrix row [last], original index is [i]
    float * sx = (float *)spriority;
    for (int idxshift = 0; idxshift < dim; idxshift += blockDim.x)
    {
        int idx = idxshift + threadIdx.x;
        if (idx < dim)
            //xi[idx] = xT[num_vec_aligned * idx + i];
            sx[idx] = x[dim_aligned * i + idx];
    }
    __syncthreads();
    while (j < num_vec)
    {
        float sum = 0;
        for (int d = 0; d < dim; d++)
        {
            //float diff = xi[d] - x[dim_aligned * j + d];
            float diff = sx[d] - xT[num_vec_aligned * d + j];
            //float diff = x[dim_aligned * i + d] - x[dim_aligned * j + d];
            //float diff = xT[num_vec_aligned * d + i] - xT[num_vec_aligned * d + j];
            sum += diff * diff;
        }
        K[num_vec_aligned * last + j] = expf(-gamma * sum);
        j += gridDim.x * blockDim.x;
    }
}

void OrcusSvmTrain(float * alpha, float * rho, const float * x, const float * y, size_t num_vec, size_t num_vec_aligned, size_t dim, size_t dim_aligned, float C, float gamma, float eps)
{
    float *d_alpha = nullptr,
        *d_x = nullptr,
        *d_xT = nullptr,
        *d_y = nullptr,
        *d_g = nullptr,
        *d_K = nullptr,
        *d_KDiag = nullptr,
        *d_reduceval = nullptr,
        *d_reduceval2 = nullptr;
    int *d_reduceidx = nullptr,
        *d_reduceidx2 = nullptr;
    float *d_lambda = nullptr;
    int *d_workingset = nullptr,
        *d_KCacheRemapIdx = nullptr,
        *d_KCacheRowIdx = nullptr,  // items at index [cache_rows] and [cache_rows+1] are indices of last inserted item
        *d_KCacheRowPriority = nullptr;  // the higher the priority is, the later was the item added

    size_t reduce_block_size = 256;
    size_t reduce_buff_size = rounduptomult(num_vec, reduce_block_size);
    size_t ones_size = std::max(num_vec_aligned, dim_aligned);
    size_t cache_size_mb = 2000;
    size_t cache_rows = cache_size_mb * 1024 * 1024 / (num_vec_aligned * sizeof(float));
    cache_rows = std::min(cache_rows, num_vec);

    std::cout << "Cache size: " << cache_rows << " rows (" << (100.f * cache_rows / (float)num_vec) << " % of data set)" << std::endl;

    hipblasHandle_t cublas;
    assert_cublas(hipblasCreate(&cublas));

    assert_cuda(hipMalloc(&d_alpha, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_x, num_vec_aligned * dim_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_xT, num_vec_aligned * dim_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_y, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_g, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceval, reduce_buff_size * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceidx, reduce_buff_size * sizeof(int)));
    assert_cuda(hipMalloc(&d_reduceval2, reduce_buff_size / reduce_block_size * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceidx2, reduce_buff_size / reduce_block_size * sizeof(int)));
    assert_cuda(hipMalloc(&d_lambda, sizeof(float)));
    assert_cuda(hipMalloc(&d_workingset, 2 * sizeof(int)));
    assert_cuda(hipMalloc(&d_KCacheRemapIdx, num_vec * sizeof(int)));
    assert_cuda(hipMalloc(&d_KCacheRowIdx, (cache_rows + 2) * sizeof(int)));  //last 2 items are indices of last cache row
    assert_cuda(hipMalloc(&d_KCacheRowPriority, cache_rows * sizeof(int)));
    assert_cuda(hipMalloc(&d_KDiag, num_vec * sizeof(float)));
    assert_cuda(hipMalloc(&d_K, cache_rows * num_vec_aligned * sizeof(float)));

    assert_cuda(hipMemset(d_alpha, 0, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMemcpy(d_x, x, num_vec_aligned * dim_aligned * sizeof(float), hipMemcpyHostToDevice));
    assert_cuda(hipMemcpy(d_y, y, num_vec_aligned * sizeof(float), hipMemcpyHostToDevice));

    float a = 1, b = 0;
    assert_cublas(hipblasSgeam(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, num_vec, dim, &a, d_x, dim_aligned, &b, d_x, num_vec_aligned, d_xT, num_vec_aligned));

    //export_cuda_buffer(d_x, dim_aligned, num_vec_aligned, sizeof(float), "x.dat");

    //dim3 dimBlock(256);
    //dim3 dimGrid(getgriddim(num_vec_aligned, (size_t)dimBlock.x));
    //kernelInitg<<<dimGrid, dimBlock>>>(d_g, num_vec_aligned);
    memsetCuda<float>(d_g, 1, num_vec_aligned);
    memsetCuda<int>(d_KCacheRemapIdx, -1, num_vec);
    memsetCuda<int>(d_KCacheRowIdx, -1, cache_rows + 2);
    memsetCuda<int>(d_KCacheRowPriority, -1, cache_rows);
    int cacheUpdateCnt = 0;
    assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_cacheUpdateCnt), &cacheUpdateCnt, sizeof(int), 0));

    //export_cuda_buffer(d_g, num_vec_aligned, 1, sizeof(float), "g.dat");

    //hipEvent_t evstart, evend;
    //hipEventCreate(&evstart);
    //hipEventCreate(&evend);
    //hipEventRecord(evstart);

    computeKDiag(d_KDiag, num_vec);
    //computeK(d_K, d_x, gamma, num_vec, num_vec_aligned, dim, dim_aligned, cublas);

    //hipEventRecord(evend);
    //assert_cuda(hipDeviceSynchronize());
    //float t;
    //hipEventElapsedTime(&t, evstart, evend);
    //std::cout << "Kernel time: " << t << std::endl;
    //hipEventDestroy(evstart);
    //hipEventDestroy(evend);

    //export_cuda_buffer(d_K, num_vec_aligned, num_vec_aligned, sizeof(float), "K.dat");

    int cacheLastPtrIdx = 0;
    dim3 dimBlock(reduce_block_size);
    dim3 dimGrid(getgriddim(num_vec_aligned, (size_t)dimBlock.x));
    dim3 dimBlockCache(256);
    dim3 dimGridCache(getgriddim(num_vec_aligned, (size_t)dimBlockCache.x));
    size_t kernelCheckCacheSMSize = std::max(dim * sizeof(float), dimBlockCache.x * sizeof(int2));
    for (int iter = 0;; iter++)
    {
        kernelSelectI<<<dimGrid, dimBlock>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec);
        //export_cuda_buffer(d_reduceval, 1, reduce_buff_size, sizeof(float), "reduceval.dat");
        //export_cuda_buffer(d_reduceidx, 1, reduce_buff_size, sizeof(int), "reduceidx.dat");
        reduceMaxIdx(d_reduceval, d_reduceidx, d_reduceval2, d_reduceidx2, num_vec_aligned, reduce_block_size);
        assert_cuda(hipMemcpy(d_workingset, d_reduceidx, sizeof(int), hipMemcpyDeviceToDevice));

        //int * KCacheRowIdx = new int[cache_rows + 2];
        //assert_cuda(hipMemcpy(KCacheRowIdx, d_KCacheRowIdx, (cache_rows + 2) * sizeof(int), hipMemcpyDeviceToHost));
        //std::cout << "KCacheRowIdx: ";
        //for (int k = 0; k < cache_rows + 2; k++)
        //    std::cout << KCacheRowIdx[k] << ", ";
        //std::cout << std::endl;
        //delete[] KCacheRowIdx;

        //check if I is cached
        kernelCheckCache<<<dimGridCache, dimBlockCache, kernelCheckCacheSMSize>>>(d_workingset, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows, d_x, d_xT, gamma, num_vec, num_vec_aligned, dim, dim_aligned, cacheLastPtrIdx);
        cacheLastPtrIdx = 1 - cacheLastPtrIdx;

        //int * KCacheRowIdx = new int[cache_rows + 2];
        //assert_cuda(hipMemcpy(KCacheRowIdx, d_KCacheRowIdx, (cache_rows + 2) * sizeof(int), hipMemcpyDeviceToHost));
        //std::cout << "KCacheRowIdx: ";
        //for (int k = 0; k < cache_rows + 2; k++)
        //    std::cout << KCacheRowIdx[k] << ", ";
        //std::cout << std::endl;
        //delete[] KCacheRowIdx;

        //kernelSelectJ1<<<dimGrid, dimBlock>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec);
        kernelSelectJCached<<<dimGrid, dimBlock>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec, num_vec_aligned, d_workingset, d_K, d_KDiag, d_KCacheRemapIdx);
        //export_cuda_buffer(d_reduceval, 1, reduce_buff_size, sizeof(float), "reduceval.dat");
        //export_cuda_buffer(d_reduceidx, 1, reduce_buff_size, sizeof(int), "reduceidx.dat");
        reduceMaxIdx(d_reduceval, d_reduceidx, d_reduceval2, d_reduceidx2, num_vec_aligned, reduce_block_size);
        assert_cuda(hipMemcpy(d_workingset + 1, d_reduceidx, sizeof(int), hipMemcpyDeviceToDevice));

        //check if J is cached
        kernelCheckCache<<<dimGridCache, dimBlockCache, kernelCheckCacheSMSize>>>(d_workingset + 1, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows, d_x, d_xT, gamma, num_vec, num_vec_aligned, dim, dim_aligned, cacheLastPtrIdx);
        cacheLastPtrIdx = 1 - cacheLastPtrIdx;
        //workaround if caching J deleted I out of cache
        kernelCheckCache<<<dimGridCache, dimBlockCache, kernelCheckCacheSMSize>>>(d_workingset, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows, d_x, d_xT, gamma, num_vec, num_vec_aligned, dim, dim_aligned, cacheLastPtrIdx);
        cacheLastPtrIdx = 1 - cacheLastPtrIdx;

        //int * KCacheRowIdx = new int[cache_rows + 2];
        //assert_cuda(hipMemcpy(KCacheRowIdx, d_KCacheRowIdx, (cache_rows + 2) * sizeof(int), hipMemcpyDeviceToHost));
        //std::cout << "KCacheRowIdx: ";
        //for (int k = 0; k < cache_rows + 2; k++)
        //    std::cout << KCacheRowIdx[k] << ", ";
        //std::cout << std::endl;
        //delete[] KCacheRowIdx;

        if (iter > 0 && iter % 1000 == 0)
        {
            int ws[2];
            float yi, yj, gi, gj;
            assert_cuda(hipMemcpy(&ws, d_workingset, 2 * sizeof(int), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&yi, d_y + ws[0], sizeof(float), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&yj, d_y + ws[1], sizeof(float), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&gi, d_g + ws[0], sizeof(float), hipMemcpyDeviceToHost));
            assert_cuda(hipMemcpy(&gj, d_g + ws[1], sizeof(float), hipMemcpyDeviceToHost));
            float diff = yi * gi - yj * gj;
            std::cout << "Iter " << iter << ": " << diff << " [" << ws[0] << "," << ws[1] << "]" <<std::endl;
            if (diff < eps)
            {
                *rho = -(yi * gi + yj * gj) / 2;
                std::cout << "Optimality reached, stopping loop. rho = " << *rho << std::endl;
                break;
            }
        }

        //kernelUpdateAlphaAndLambda<<<1, 1>>>(d_alpha, d_lambda, d_y, d_g, d_K, C, d_workingset, num_vec, num_vec_aligned);
        //kernelUpdateg<<<dimGrid, dimBlock>>>(d_g, d_lambda, d_y, d_K, d_workingset, num_vec, num_vec_aligned);
        kernelUpdateAlphaAndLambdaCached<<<1, 1>>>(d_alpha, d_lambda, d_y, d_g, d_K, C, d_workingset, num_vec, num_vec_aligned, d_KDiag, d_KCacheRemapIdx);
        kernelUpdategCached<<<dimGrid, dimBlock>>>(d_g, d_lambda, d_y, d_K, d_workingset, num_vec, num_vec_aligned, d_KCacheRemapIdx);

        //float lambda;
        //int ws[2];
        //assert_cuda(hipMemcpy(&lambda, d_lambda, sizeof(float), hipMemcpyDeviceToHost));
        //assert_cuda(hipMemcpy(&ws, d_workingset, 2 * sizeof(int), hipMemcpyDeviceToHost));
        //std::cout << "i: " << ws[0] << ", j: " << ws[1] << ", lambda: " << lambda << std::endl;
    }

    assert_cuda(hipMemcpyFromSymbol(&cacheUpdateCnt, HIP_SYMBOL(d_cacheUpdateCnt), sizeof(int), 0));
    std::cout << "Cache row updates: " << cacheUpdateCnt << std::endl;

    assert_cuda(hipMemcpy(alpha, d_alpha, num_vec * sizeof(float), hipMemcpyDeviceToHost));

    assert_cuda(hipFree(d_K));
    assert_cuda(hipFree(d_KDiag));
    assert_cuda(hipFree(d_KCacheRemapIdx));
    assert_cuda(hipFree(d_KCacheRowIdx));
    assert_cuda(hipFree(d_KCacheRowPriority));
    assert_cuda(hipFree(d_alpha));
    assert_cuda(hipFree(d_x));
    assert_cuda(hipFree(d_xT));
    assert_cuda(hipFree(d_y));
    assert_cuda(hipFree(d_g));
    assert_cuda(hipFree(d_reduceval));
    assert_cuda(hipFree(d_reduceidx));
    assert_cuda(hipFree(d_reduceval2));
    assert_cuda(hipFree(d_reduceidx2));
    assert_cuda(hipFree(d_lambda));
    assert_cublas(hipblasDestroy(cublas));
}