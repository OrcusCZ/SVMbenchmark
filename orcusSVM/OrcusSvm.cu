#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "OrcusSvm.h"
#include "cudaerror.cuh"
#include "debug.h"

template<typename T>
T getgriddim(T totallen, T blockdim)
{
    return (totallen + blockdim - (T)1) / blockdim;
}

template<typename T>
T rounduptomult(T x, T m)
{
    return ((x + m - (T)1) / m) * m;
}

__global__ void kernelInitg(float * g, int len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len)
        g[i] = 1;
}

__global__ void kernelComputeK(float * K, const float * x, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x,
        j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < num_vec && j < num_vec)
    {
        float sum = 0;
        for (int d = 0; d < dim; d++)
        {
            float diff = x[dim_aligned * i + d] - x[dim_aligned * j + d];
            sum += diff * diff;
        }
        K[num_vec_aligned * j + i] = exp(-gamma * sum);
    }
#ifdef _DEBUG
    else
        K[num_vec_aligned * j + i] = 0;
#endif
}

__global__ void kernelSelectI(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < num_vec)
    {
        float y_ = y[i];
        float a_ = alpha[i];
        if ((y_ == 1 && a_ < C) || (y_ == -1 && a_ > 0))
            valbuf[i] = y[i] * g[i];
        else
            valbuf[i] = -FLT_MAX;
        idxbuf[i] = i;
    }
    else
        valbuf[i] = -FLT_MAX;
}

//first order search
__global__ void kernelSelectJ1(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < num_vec)
    {
        float y_ = y[i];
        float a_ = alpha[i];
        if ((y_ == 1 && a_ > 0) || (y_ == -1 && a_ < C))
            valbuf[i] = -y[i] * g[i]; //return negative, so we can use reducemax
        else
            valbuf[i] = -FLT_MAX;
        idxbuf[i] = i;
    }
    else
        valbuf[i] = -FLT_MAX;
}

__global__ void kernelReduceMaxIdxInplace(float * val, int * idx, int len)
{
    extern __shared__ float sval[];
    int * sidx = (int *)(sval + blockDim.x);

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len)
    {
        sval[threadIdx.x] = val[i];
        sidx[threadIdx.x] = idx[i];
    }
    else
    {
        sval[threadIdx.x] = -FLT_MAX;
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (sval[threadIdx.x + s] > sval[threadIdx.x])
            {
                sval[threadIdx.x] = sval[threadIdx.x + s];
                sidx[threadIdx.x] = sidx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        val[blockIdx.x] = sval[0];
        idx[blockIdx.x] = sidx[0];
    }
}

void reduceMaxIdxInplace(float * d_val, int * d_idx, int len, int reduce_block_size)
{
    //int orig_len = len;
    dim3 dimBlock = dim3(reduce_block_size);
    while (len > 1)
    {
        dim3 dimGrid = dim3(getgriddim(len, (int)dimBlock.x));
        kernelReduceMaxIdxInplace<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val, d_idx, len);
        len = dimGrid.x;
    }
    //export_cuda_buffer(d_val, 1, orig_len, sizeof(float), "reduceval.dat");
    //export_cuda_buffer(d_idx, 1, orig_len, sizeof(int), "reduceidx.dat");
}

__global__ void kernelComputeLambda(float * lambda, const float * y, const float * g, const float * K, const float * alpha, float C, int i, int j, int num_vec_aligned)
{
    float l1 = y[i] == 1 ? C - alpha[i] : alpha[i];
    float l2 = y[j] == 1 ? alpha[j] : C - alpha[j];
    float l3 = (y[i] * g[i] - y[j] * g[j]) / (K[num_vec_aligned * i + i] + K[num_vec_aligned * j + j] - 2 * K[num_vec_aligned * i + j]);
    *lambda = min(l1, min(l2, l3));
}

__global__ void kernelUpdateg(float * g, const float * lambda, const float * y, const float * K, int i, int j, int num_vec, int num_vec_aligned)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        g[k] += *lambda * y[k] * (K[num_vec_aligned * j + k] - K[num_vec_aligned * i + k]);
    }
}

__global__ void kernelUpdateAlpha(float * alpha, const float * lambda, const float * y, int i, int j, int num_vec)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        float l = *lambda;
        alpha[i] += l * y[i];
        alpha[j] -= l * y[j];
    }
}

void OrcusSvmTrain(float * alpha, float * rho, const float * x, const float * y, size_t num_vec, size_t num_vec_aligned, size_t dim, size_t dim_aligned, float C, float gamma, float eps)
{
    float *d_alpha = nullptr,
        *d_x = nullptr,
        *d_y = nullptr,
        *d_g = nullptr,
        *d_K = nullptr,
        *d_reduceval = nullptr;
    int *d_reduceidx = nullptr;
    float *d_lambda = nullptr;

    size_t reduce_block_size = 256;
    size_t reduce_buff_size = rounduptomult(num_vec, reduce_block_size);

    assert_cuda(hipMalloc(&d_alpha, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_x, num_vec_aligned * dim_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_y, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_g, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_K, num_vec_aligned * num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceval, reduce_buff_size * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceidx, reduce_buff_size * sizeof(int)));
    assert_cuda(hipMalloc(&d_lambda, sizeof(float)));

    assert_cuda(hipMemset(d_alpha, 0, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMemcpy(d_x, x, num_vec_aligned * dim_aligned * sizeof(float), hipMemcpyHostToDevice));
    assert_cuda(hipMemcpy(d_y, y, num_vec_aligned * sizeof(float), hipMemcpyHostToDevice));

    export_cuda_buffer(d_x, dim_aligned, num_vec_aligned, sizeof(float), "x.dat");

    dim3 dimBlock(256);
    dim3 dimGrid(getgriddim(num_vec_aligned, (size_t)dimBlock.x));
    kernelInitg<<<dimGrid, dimBlock>>>(d_g, num_vec_aligned);

    export_cuda_buffer(d_g, num_vec_aligned, 1, sizeof(float), "g.dat");

    dimBlock = dim3(16, 16);
    dimGrid = dim3(getgriddim(num_vec_aligned, (size_t)dimBlock.x), getgriddim(num_vec_aligned, (size_t)dimBlock.y));
    kernelComputeK<<<dimGrid, dimBlock>>>(d_K, d_x, gamma, num_vec, num_vec_aligned, dim, dim_aligned);

    export_cuda_buffer(d_K, num_vec_aligned, num_vec_aligned, sizeof(float), "K.dat");

    for (int iter = 0; iter < 20; iter++)
    {
        dimBlock = dim3(reduce_block_size);
        dimGrid = dim3(getgriddim(num_vec_aligned, (size_t)dimBlock.x));

        kernelSelectI<<<dimGrid, dimBlock>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec);
        //export_cuda_buffer(d_reduceval, 1, reduce_buff_size, sizeof(float), "reduceval.dat");
        //export_cuda_buffer(d_reduceidx, 1, reduce_buff_size, sizeof(int), "reduceidx.dat");
        reduceMaxIdxInplace(d_reduceval, d_reduceidx, num_vec_aligned, reduce_block_size);
        int ws_i;
        assert_cuda(hipMemcpy(&ws_i, d_reduceidx, sizeof(int), hipMemcpyDeviceToHost));

        kernelSelectJ1<<<dimGrid, dimBlock>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec);
        //export_cuda_buffer(d_reduceval, 1, reduce_buff_size, sizeof(float), "reduceval.dat");
        //export_cuda_buffer(d_reduceidx, 1, reduce_buff_size, sizeof(int), "reduceidx.dat");
        reduceMaxIdxInplace(d_reduceval, d_reduceidx, num_vec_aligned, reduce_block_size);
        int ws_j;
        assert_cuda(hipMemcpy(&ws_j, d_reduceidx, sizeof(int), hipMemcpyDeviceToHost));

        std::cout << "Found working set pair: " << ws_i << ", " << ws_j << "; ";

        kernelComputeLambda<<<1, 1>>>(d_lambda, d_y, d_g, d_K, d_alpha, C, ws_i, ws_j, num_vec_aligned);
        float lambda;
        assert_cuda(hipMemcpy(&lambda, d_lambda, sizeof(float), hipMemcpyDeviceToHost));
        std::cout << "Lambda: " << lambda << std::endl;

        kernelUpdateg<<<dimGrid, dimBlock>>>(d_g, d_lambda, d_y, d_K, ws_i, ws_j, num_vec, num_vec_aligned);
        kernelUpdateAlpha<<<1, 1>>>(d_alpha, d_lambda, d_y, ws_i, ws_j, num_vec);
    }

    assert_cuda(hipFree(d_alpha));
    assert_cuda(hipFree(d_x));
    assert_cuda(hipFree(d_y));
    assert_cuda(hipFree(d_g));
    assert_cuda(hipFree(d_K));
    assert_cuda(hipFree(d_reduceval));
    assert_cuda(hipFree(d_reduceidx));
    assert_cuda(hipFree(d_lambda));
}