#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <ctime>
#include <algorithm>
#include <vector>
#include <numeric>
#include <cfloat>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "OrcusSvm.h"
#include "../cudaerror.h"
#include "../cuda_utils.h"
#include "csr.h"
#include "calc_x2.h"

#if __cplusplus <= 199711L
#define nullptr NULL
#endif

//#define USE_TIMERS
#include "../debug.h"

#define CALC_KLOCAL
#define USE_CUBLAS
//#define USE_DAIFLETCHER
#define MAX_BLOCK_ITER 10000

#define TRANSPOSE_TILE_SIZE 16
#define DENSE_TILE_SIZE 16

#define MAX_WORKING_SET 2048

#define NUM_SORT_BLOCKS 32
#define CALC_KLOCAL_TILE_X 16
#define CALC_KLOCAL_NUM_WARPS 8

//#define WORKING_SET 2048
//#define NUM_NC (WORKING_SET/4)
//#define ELEM_PER_THREAD 2

//#define WORKING_SET 1024
//#define NUM_NC (WORKING_SET/4)
//#define ELEM_PER_THREAD 1

namespace OrcusSVM1B
{
#include "dev_vars.h"
}

using namespace OrcusSVM1B;

#include "check_cache.h"
#include "kernels_select_ws.h"
#include "find_nbestv2.h"
#include "kernels_klocal.h"
#ifdef USE_DAIFLETCHER
#include "kernels_daifletcher.h"
#else
#include "kernels_smo.h"
#endif

extern int g_cache_size;

#define STATIC_MIN(a, b) ((a) < (b) ? (a) : (b))
#define STATIC_MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ static void kernelTranspose(const float * data_in, float * data_out, int width, int height, int ipitch, int opitch)
{
	__shared__ float tile[TRANSPOSE_TILE_SIZE][TRANSPOSE_TILE_SIZE + 1];
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x,
		yIndexO = blockDim.x * blockIdx.x + threadIdx.y;
    for (int offset = blockDim.y * blockIdx.y; offset < height; offset += gridDim.y * blockDim.y)
	{
        int yIndex = offset + threadIdx.y;
        int xIndexO = offset + threadIdx.x;
		if (xIndex < width && yIndex < height)
			tile[threadIdx.y][threadIdx.x] = data_in[ipitch * yIndex + xIndex];
		__syncthreads();
		if (xIndexO < height && yIndexO < width)
			data_out[opitch * yIndexO + xIndexO] = tile[threadIdx.x][threadIdx.y];
		__syncthreads();
	}
}

static void computeKDiag(float * d_KDiag, int num_vec)
{
    //K[i,i] is always 1 for RBF kernel, let's just use memset here
    memsetCuda<float>(d_KDiag, 1, num_vec);
}

template<unsigned int WS>
__global__ static void kernelKtoHLocal(float * K, const float * y, const int * ws)
{
    for (int iy = blockDim.y * blockIdx.y + threadIdx.y; iy < WS; iy += gridDim.y * blockDim.y)
    {
        float yy = y[ws[iy]];
        for (int ix = blockDim.x * blockIdx.x + threadIdx.x; ix < WS; ix += gridDim.x * blockDim.x)
            K[WS * iy + ix] *= yy * y[ws[ix]];
    }
}

template<unsigned int WS>
__global__ static void kernelUpdateG(float * y, float * g, const float * alphadiff, const int * ws, const float * K, const int * KCacheRemapIdx, int num_vec, int num_vec_aligned)
{
    __shared__ float shAdiff[WS];
    __shared__ int shWS[WS];
    __shared__ float shY[WS];
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    for (int j = threadIdx.x; j < WS; j += blockDim.x) 
    {
        shAdiff[j] = alphadiff[j];
        shWS[j] = KCacheRemapIdx[ws[j]];
        shY[j] = y[ws[j]];
        if (blockIdx.x == 0)
        {
            if (shAdiff[j] != 0) //use some tau here
                atomicAdd(d_updateGCnt, 1);
            else
                atomicAdd(d_updateGCnt + 1, 1);
        }
    }
    __syncthreads();

    if (k < num_vec)
    {
#ifdef USE_DAIFLETCHER
        float update = 0;
        for (int i = 0; i < WS; i++)
        {
            float adiff = shAdiff[i];
			if (adiff != 0) 
                update += adiff * /*shY[i] **/ K[(size_t)num_vec_aligned * shWS[i] + k];
        }
        g[k] += update;
#else
        float update = 0;
        for (int i = 0; i < WS; i++)
        {
            float adiff = shAdiff[i];
			if (adiff != 0) 
                update += adiff * K[(size_t)num_vec_aligned * shWS[i] + k];
        }
        g[k] += y[k] * update;
#endif
    }
}

template<unsigned int WS>
__global__ static void kernelUpdateGHLocal(float * gh, const float * g, const float * y, const float * alpha, const int * ws, const float * K)
{
    __shared__ float shAlpha[WS];
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    int ws_ = ws[k];
    shAlpha[k] = alpha[ws_];
    __syncthreads();

    float aux = 0;
    for (int i = 0; i < WS; i++)
        aux += shAlpha[i] * K[WS * i + k];
    gh[k] = g[ws_] * y[ws_] - 1.0f - aux;
}

template<unsigned int WS>
static void train(float * alpha, float * rho, bool sparse, const OrcusSvm1B::Data & x, const float * y, size_t num_vec, size_t num_vec_aligned, size_t dim, size_t dim_aligned, float C, float gamma, float eps)
{
    const int NC = WS / 4;
    const int ELEM_PER_THREAD = STATIC_MAX(WS / 1024, 1);
    std::cout << "Working set: " << WS
        << "\nNC: " << NC
        << "\nElements per thread: " << ELEM_PER_THREAD << std::endl;
    assert(WS <= MAX_WORKING_SET);

    //bool use_ellpack = dim <= 1000;
    bool use_ellpack = false;
    //num_vec = 1000;
    //num_vec_aligned = 1024;
    float *d_alpha = nullptr,
        *d_alphadiff = nullptr,
        *d_x = nullptr,
#ifdef USE_CUBLAS
        *d_xT = nullptr,
#endif
        *d_y = nullptr,
        *d_g = nullptr,
        *d_K = nullptr,
        *d_KLocal = nullptr,
        *d_KDiag = nullptr,
        *d_sortval = nullptr;
    int *d_sortidx = nullptr,
        *d_workingset = nullptr,
        *d_ws_priority = nullptr,
        *d_KCacheRemapIdx = nullptr,
        *d_KCacheRowIdx = nullptr,  // items at index [cache_rows] and [cache_rows+1] are indices of last inserted item
        *d_KCacheRowPriority = nullptr;  // the higher the priority is, the later was the item added
    float *d_denseVec = nullptr;  //dense vector used to calculate K cache row for sparse data
    float *d_x2 = nullptr;
	float *d_sortValues = nullptr;
	int *d_sortIdxs = nullptr;
#ifdef USE_CUBLAS
    float * d_xTile = nullptr;
    float * d_KTile = nullptr;
#endif
#ifdef USE_DAIFLETCHER
    float * d_df_g = nullptr,
          * d_df_gh = nullptr,
          * d_df_y = nullptr,
          * d_df_tempv = nullptr,
          * d_df_d = nullptr,
          * d_df_Ad = nullptr,
          * d_df_t = nullptr,
          * d_df_xplus = nullptr,
          * d_df_tplus = nullptr,
          * d_df_sk = nullptr,
          * d_df_yk = nullptr;
#endif
    csr_gpu sparse_data_gpu;
    ellpack_gpu ellpack_data_gpu;
    jds_gpu jds_data_gpu;
    hipblasHandle_t cublas;
    try
    {

        std::cout << "Training data: " << (sparse ? "sparse" : "dense") << std::endl;
        std::cout << "Data size: " << num_vec << "\nDimension: " << dim << std::endl;

        float aux_one = 1,
            aux_zero = 0;
        assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_aux_one), &aux_one, sizeof(aux_one)));
        assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_aux_zero), &aux_zero, sizeof(aux_zero)));
        
        assert_cublas(hipblasCreate(&cublas));
        
        assert_cuda(hipMalloc(&d_x2, num_vec * sizeof(float)));
        assert_cuda(hipMalloc(&d_alpha, num_vec_aligned * sizeof(float)));
        assert_cuda(hipMalloc(&d_alphadiff, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_y, num_vec_aligned * sizeof(float)));
        assert_cuda(hipMalloc(&d_g, num_vec_aligned * sizeof(float)));
        assert_cuda(hipMalloc(&d_sortval, num_vec * sizeof(float)));
        assert_cuda(hipMalloc(&d_sortidx, num_vec * sizeof(int)));
        assert_cuda(hipMalloc(&d_workingset, WS * sizeof(int)));
        assert_cuda(hipMalloc(&d_ws_priority, num_vec * sizeof(int)));
        assert_cuda(hipMalloc(&d_KCacheRemapIdx, num_vec * sizeof(int)));
        //assert_cuda(hipMalloc(&d_KCacheRowIdx, cache_rows * sizeof(int)));
        //assert_cuda(hipMalloc(&d_KCacheRowPriority, cache_rows * sizeof(int)));
        assert_cuda(hipMalloc(&d_KDiag, num_vec * sizeof(float)));
        //assert_cuda(hipMalloc(&d_K, cache_rows * num_vec_aligned * sizeof(float)));
        assert_cuda(hipMalloc(&d_KLocal, WS * WS * sizeof(float)));

        //assert_cuda(hipMalloc(&d_sortValues, NUM_SORT_BLOCKS * WS * sizeof(float)));
        //assert_cuda(hipMalloc(&d_sortIdxs, NUM_SORT_BLOCKS * WS * sizeof(int)));
        assert_cuda(hipMalloc(&d_sortValues, NUM_SORT_BLOCKS * NC * 2 * sizeof(float)));
        assert_cuda(hipMalloc(&d_sortIdxs, NUM_SORT_BLOCKS * NC * 2 * sizeof(int)));

        assert_cuda(hipMemset(d_alpha, 0, num_vec_aligned * sizeof(float)));
        if (sparse)
        {
            if (use_ellpack)
                makeCudaEllpack(ellpack_data_gpu, *x.sparse);
            else
                makeCudaCsr(sparse_data_gpu, *x.sparse);
            makeCudaJds(jds_data_gpu, *x.sparse);
            assert_cuda(hipMalloc(&d_denseVec, dim_aligned * WS * sizeof(float)));
            std::cout << "Precalculating X2" << std::endl;
            //if (use_ellpack)
            //    computeX2Sparse(d_x2, ellpack_data_gpu, num_vec);
            //else
            //    computeX2Sparse(d_x2, sparse_data_gpu, num_vec);
            computeX2Sparse(d_x2, jds_data_gpu, num_vec);
        }
        else
        {
            assert_cuda(hipMalloc(&d_x, num_vec_aligned * dim_aligned * sizeof(float)));
            assert_cuda(hipMemcpy(d_x, x.dense, num_vec_aligned * dim_aligned * sizeof(float), hipMemcpyHostToDevice));
            std::cout << "Precalculating X2" << std::endl;
            computeX2Dense(d_x2, d_x, num_vec, num_vec_aligned, dim, dim_aligned);
#ifdef USE_CUBLAS
            assert_cuda(hipMalloc(&d_xT, dim_aligned * num_vec_aligned * sizeof(*d_xT)));
            assert_cuda(hipMalloc(&d_xTile, dim_aligned * WS * sizeof(float)));
            assert_cuda(hipMalloc(&d_KTile, num_vec_aligned * WS * sizeof(float)));
            dim3 dimBlockT(TRANSPOSE_TILE_SIZE, TRANSPOSE_TILE_SIZE);
            kernelTranspose << <dim3(getgriddim<int>(dim, dimBlockT.x), 16), dimBlockT >> >(d_x, d_xT, dim, num_vec, dim_aligned, num_vec_aligned);
#endif
        }
        assert_cuda(hipMemcpy(d_y, y, num_vec_aligned * sizeof(float), hipMemcpyHostToDevice));

        size_t cache_size_mb = g_cache_size;
        if (cache_size_mb == 0)  //TODO: move cache size calculation after all allocations
        {
            size_t free_mem, total_mem;
            assert_cuda(hipFree(nullptr));  //force CUDA init
            assert_cuda(hipMemGetInfo(&free_mem, &total_mem));
            cache_size_mb = (free_mem * 0.8) / (1024 * 1024);
        }
        size_t cache_rows = cache_size_mb * 1024 * 1024 / (num_vec_aligned * sizeof(float));
        cache_rows = std::min(cache_rows, num_vec);
        std::cout << "Cache size: " << cache_rows << " rows (" << (100.f * cache_rows / (float)num_vec) << " % of data set)" << std::endl;
        if (cache_rows < WS)
        {
            std::cout << "Cache smaller than working set, can't continue" << std::endl;
            return;
        }

        assert_cuda(hipMalloc(&d_KCacheRowIdx, cache_rows * sizeof(int)));
        assert_cuda(hipMalloc(&d_KCacheRowPriority, cache_rows * sizeof(int)));
        assert_cuda(hipMalloc(&d_K, cache_rows * num_vec_aligned * sizeof(float)));

#ifdef USE_DAIFLETCHER
        assert_cuda(hipMalloc(&d_df_g, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_gh, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_y, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_tempv, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_d, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_Ad, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_t, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_xplus, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_tplus, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_sk, WS * sizeof(float)));
        assert_cuda(hipMalloc(&d_df_yk, WS * sizeof(float)));
#endif

        memsetCuda<int>(d_ws_priority, 0, num_vec);
#ifdef USE_DAIFLETCHER
        assert_cuda(hipMemset(d_g, 0, num_vec_aligned * sizeof(float)));
#else
        memsetCuda<float>(d_g, 1, num_vec_aligned);
#endif
        memsetCuda<int>(d_KCacheRemapIdx, -1, num_vec);
        memsetCuda<int>(d_KCacheRowIdx, -1, cache_rows);
        memsetCuda<int>(d_KCacheRowPriority, -1, cache_rows);
        int cacheUpdateCnt = 0;
        assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_cacheUpdateCnt), &cacheUpdateCnt, sizeof(int), 0));
        int cacheChanges[6] = { -1, -1, -1, -1, -1, -1 };
        assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_KCacheChanges), cacheChanges, 3 * sizeof(int2), 0));

        std::cout << "Precalculating KDiag" << std::endl;
        computeKDiag(d_KDiag, num_vec);

        size_t num_vec_shrunk = num_vec;

        thrust::device_ptr<float> dev_sortval = thrust::device_pointer_cast(d_sortval);
        thrust::device_ptr<int> dev_sortidx = thrust::device_pointer_cast(d_sortidx);

        const int findActiveSetBlockSize = STATIC_MAX(128, NC);
        const int findCacheRowBlockSize = STATIC_MAX(512, WS / ELEM_PER_THREAD);
        const int fillWSBlockSize = STATIC_MIN(512, WS);
        dim3 dimBlock(256);
        dim3 dimGrid(getgriddim(num_vec, (size_t)dimBlock.x));
        dim3 dimBlock32x8(32, 8);
        dim3 dimBlockFindActiveSet(findActiveSetBlockSize);
        dim3 dimGridFindActiveSet(std::min<int>(dimBlockFindActiveSet.x, WS));
        dim3 dimBlockCalcKLocal(32, CALC_KLOCAL_NUM_WARPS);
        dim3 dimGridCalcKLocal(WS / (CALC_KLOCAL_TILE_X * dimBlockCalcKLocal.y), WS);
        size_t total_num_iter = 0;
        assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_total_num_iter), &total_num_iter, sizeof(total_num_iter)));
        float last_diff = 0;

        //kernel grid/block dimension check
        //kernelFindNBest
        assert(findActiveSetBlockSize >= NC && NUM_SORT_BLOCKS * NC > NC + findActiveSetBlockSize);
        //check cache
        assert(findCacheRowBlockSize * ELEM_PER_THREAD >= WS);

        //kernel timers
        float timer_check_cache = 0;
        int counter_check_cache = 0;
        float timer_local_solver = 0;
        int counter_local_solver = 0;
        float timer_Gupdate = 0;
        int counter_Gupdate = 0;
        float timer_find_active_set = 0;
        int counter_find_active_set = 0;
        float timer_find_nbest = 0;
        int counter_find_nbest = 0;
        float timer_fill_ws = 0;
        int counter_fill_ws = 0;
        float timer_calc_klocal = 0;
        int counter_calc_klocal = 0;

        int MAX_GLOBAL_ITERS = 0;
        SYNC_RESET(0);

        int updateGCnt[2] = { 0 };
        assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_updateGCnt), updateGCnt, sizeof(updateGCnt)));

        //std::ofstream fout_wsp("wsp.dat", std::ios::out | std::ios::binary);

        std::cout << "Starting iterations" << std::endl;
        for (int iter = 0;; iter++)
        {
            //std::cout << "Iter " << iter << std::endl;
#if 0
            kernelPrepareSortI<<<dimGrid, dimBlock>>>(d_sortval, d_sortidx, d_y, d_g, d_alpha, C, num_vec_shrunk);
            thrust::sort_by_key(dev_sortval, dev_sortval + num_vec, dev_sortidx, thrust::greater<float>());
            hipMemcpy(d_workingset, d_sortidx, WS / 2 * sizeof(int), hipMemcpyDeviceToDevice);
            //float mi, mj;
            //hipMemcpy(&mi, d_sortval, sizeof(float), hipMemcpyDeviceToHost);

            //for (int i = 0; i < WS / 2; i++)
            //    checkCache(sparse, d_workingset + i, d_x, d_x2, sparse_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma);

            kernelPrepareSortJ<<<dimGrid, dimBlock>>>(d_sortval, d_sortidx, d_y, d_g, d_alpha, C, num_vec_shrunk);
            //kernelPrepareSortJSecondOrder<<<dimGrid, dimBlock>>>(d_sortval, d_sortidx, d_y, d_g, d_alpha, C, num_vec_shrunk, num_vec_aligned, d_workingset, d_K, d_KDiag, d_KCacheRemapIdx);
            thrust::sort_by_key(dev_sortval, dev_sortval + num_vec, dev_sortidx, thrust::greater<float>());
            hipMemcpy(d_workingset + WS / 2, d_sortidx, WS / 2 * sizeof(int), hipMemcpyDeviceToDevice);
            //hipMemcpy(&mj, d_sortval, sizeof(float), hipMemcpyDeviceToHost);

            //for (int i = WS / 2; i < WS; i++)
            //    checkCache(sparse, d_workingset + i, d_x, d_x2, sparse_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma);
#else
            //std::cout << "gridDim: " << dimGrid32.x << ", blockDim: " << dimBlock.x << std::endl;
            //ACCUMULATE_KERNEL_TIME(timer_find_active_set, counter_find_active_set, (kernelFindActiveSet<findActiveSetBlocksize, WS><<<dimGridFindActiveSet, dimBlockFindActiveSet>>>(d_workingset, d_y, d_g, d_alpha, C, num_vec_shrunk)));
            if (iter == 0)
            {
                //SYNC_RESET(0);
                //ACCUMULATE_KERNEL_TIME(timer_find_active_set, counter_find_active_set, (kernelFindActiveSetV2<findActiveSetBlocksize, WS, NUM_SORT_BLOCKS><<<NUM_SORT_BLOCKS, findActiveSetBlocksize>>>(d_workingset, d_y, d_g, d_alpha, C, num_vec_shrunk, d_sortValues, d_sortIdxs, SYNC_BUFFER(0))));

                kernelPrepareSortI << <dimGrid, dimBlock >> >(d_sortval, d_sortidx, d_y, d_g, d_alpha, C, num_vec_shrunk);
                thrust::sort_by_key(dev_sortval, dev_sortval + num_vec, dev_sortidx, thrust::greater<float>());
                hipMemcpy(d_workingset, d_sortidx, WS / 2 * sizeof(int), hipMemcpyDeviceToDevice);
                kernelPrepareSortJ << <dimGrid, dimBlock >> >(d_sortval, d_sortidx, d_y, d_g, d_alpha, C, num_vec_shrunk);
                thrust::sort_by_key(dev_sortval, dev_sortval + num_vec, dev_sortidx, thrust::greater<float>());
                hipMemcpy(d_workingset + WS / 2, d_sortidx, WS / 2 * sizeof(int), hipMemcpyDeviceToDevice);
                /*std::vector<int> ws(WS);
                for (int i = 0; i < WS; i++)
                ws[i] = i;
                hipMemcpy(d_workingset, &ws[0], WS * sizeof(int), hipMemcpyHostToDevice);*/
            }
            else
            {
                //SYNC_RESET(0);
#if 0
                ACCUMULATE_KERNEL_TIME(timer_find_nbest, counter_find_nbest, (kernelFindNBest<findActiveSetBlockSize, NC, NUM_SORT_BLOCKS><<<NUM_SORT_BLOCKS, findActiveSetBlockSize>>>(d_workingset, d_y, d_g, d_alpha, C, num_vec_shrunk, d_ws_priority, d_sortValues, d_sortIdxs, SYNC_BUFFER(0))));
#else
                ACCUMULATE_KERNEL_TIME(timer_find_nbest, counter_find_nbest, (kernelFindNBestV2<findActiveSetBlockSize, NC, NUM_SORT_BLOCKS> << <NUM_SORT_BLOCKS, findActiveSetBlockSize >> >(d_workingset, d_y, d_g, d_alpha, C, num_vec_shrunk, d_ws_priority, d_sortValues, d_sortIdxs, SYNC_BUFFER(0))));
#endif
                //SYNC_RESET(0);
                ACCUMULATE_KERNEL_TIME(timer_fill_ws, counter_fill_ws, (kernelFillWorkingSet<WS, NC> << <1, fillWSBlockSize >> >(d_workingset, d_alpha, C, d_ws_priority, d_sortIdxs)));

                //std::vector<int> ws_priority(num_vec);
                //assert_cuda(hipMemcpy(&ws_priority[0], d_ws_priority, num_vec * sizeof(int), hipMemcpyDeviceToHost));
                //fout_wsp.write((const char *)&ws_priority[0], num_vec * sizeof(int));
            }
            //export_cuda_buffer(d_sortValues, WS, NUM_SORT_BLOCKS, sizeof(float), "sortValues.dat");
            //export_cuda_buffer(d_sortIdxs, WS, NUM_SORT_BLOCKS, sizeof(int), "sortIdxs.dat");
#endif
            //if (iter == 1)
            //import_cuda_buffer(d_workingset, WS, 1, sizeof(int), "E:\\ws_" + std::to_string(iter) + ".dat");
            //std::cout << "===\nWS: ";
            //print_cuda_buffer(d_workingset, WS, 1, std::cout);
            //int ws[WS];
            //for (int i = 0; i < WS; i++)
            //    ws[i] = i;
            //assert_cuda(hipMemcpy(d_workingset, ws, WS * sizeof(int), hipMemcpyHostToDevice));

#ifndef CALC_KLOCAL
            if (sparse)
            {
                ACCUMULATE_KERNEL_TIME(timer_check_cache, counter_check_cache, (checkCacheV2<findCacheRowBlockSize, WS, ELEM_PER_THREAD>(sparse, d_workingset, d_x, d_x2, sparse_data_gpu, ellpack_data_gpu, jds_data_gpu, use_ellpack, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma)));
            }
            else
            {
#ifdef USE_CUBLAS
                ACCUMULATE_KERNEL_TIME(timer_check_cache, counter_check_cache, (checkCacheCublas<findCacheRowBlockSize, WS, ELEM_PER_THREAD>(sparse, d_workingset, d_x, d_xT, d_xTile, d_x2, sparse_data_gpu, d_K, d_KTile, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma, cublas)));
#else
                ACCUMULATE_KERNEL_TIME(timer_check_cache, counter_check_cache, (checkCacheV2<findCacheRowBlockSize, WS, ELEM_PER_THREAD>(sparse, d_workingset, d_x, d_x2, sparse_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma)));
#endif
            }
            ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelCopyKToLocal<WS><<<WS, WS>>>(d_workingset, d_K, d_KLocal, d_KCacheRemapIdx, num_vec_aligned)));
#else
#if 0
            ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelCalcKLocal<CALC_KLOCAL_TILE_X, CALC_KLOCAL_NUM_WARPS><<<dimGridCalcKLocal, dimBlockCalcKLocal>>>(d_KLocal, d_K, d_KCacheRemapIdx, d_workingset, d_x, d_x2, gamma, num_vec_aligned, dim, dim_aligned)));
#else
            if (sparse)
            {
                assert_cuda(hipMemset(d_denseVec, 0, dim_aligned * WS * sizeof(float)));
                dim3 dimBlock(256);
                if (use_ellpack)
                {
                    dim3 dimGrid(getgriddim<int>(WS, dimBlock.x));
                    ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelMakeDenseVecWSKLocal<WS> << <dimGrid, dimBlock >> >(d_KCacheRemapIdx, ellpack_data_gpu, d_workingset, d_denseVec, dim_aligned)));
                    dimGrid = dim3(getgriddim<int>(WS, dimBlock.x), WS);
                    ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelCalcKLocalSparse<WS> << <dimGrid, dimBlock >> >(d_KLocal, d_K, d_KCacheRemapIdx, ellpack_data_gpu, d_x2, d_denseVec, d_workingset, gamma, num_vec, num_vec_aligned, dim, dim_aligned)));
                }
                else
                {
                    dim3 dimGrid(std::min(64, getgriddim<int>(dim, dimBlock.x)), WS);
                    ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelMakeDenseVecWSKLocal<WS> << <dimGrid, dimBlock >> >(d_KCacheRemapIdx, sparse_data_gpu, d_workingset, d_denseVec, dim_aligned)));
                    const int NUM_WARPS = 8;
                    dimBlock = dim3(32, NUM_WARPS);
                    dimGrid = dim3(WS / NUM_WARPS, WS);
                    ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelCalcKLocalSparse<WS, NUM_WARPS> << <dimGrid, dimBlock >> >(d_KLocal, d_K, d_KCacheRemapIdx, sparse_data_gpu, d_x2, d_denseVec, d_workingset, gamma, num_vec, num_vec_aligned, dim, dim_aligned)));
                }
            }
            else
            {
                const int BLOCK_SIZE = 16;
                dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
                dim3 dimGrid(WS / BLOCK_SIZE, WS / BLOCK_SIZE);
                //ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelCalcKLocalV2_NT<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_KLocal, d_x, d_xT, d_x2, d_workingset, gamma, num_vec, num_vec_aligned, dim, dim_aligned, WS)));
                ACCUMULATE_KERNEL_TIME(timer_calc_klocal, counter_calc_klocal, (kernelCalcKLocalV2_NN<WS, BLOCK_SIZE> << <dimGrid, dimBlock >> >(d_KLocal, d_K, d_KCacheRemapIdx, d_x, d_x2, d_y, d_workingset, gamma, num_vec_aligned, dim, dim_aligned)));
            }
#endif
            //export_cuda_buffer(d_KLocal, WS, WS, sizeof(float), "KLocal.dat");
#endif
            //for (int i = 0; i < WS; i++)
            //ACCUMULATE_KERNEL_TIME(timer_check_cache, counter_check_cache, (checkCache(sparse, d_workingset + i, d_x, d_x2, sparse_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma)));
            //export_cuda_buffer(d_K, num_vec_aligned, cache_rows, sizeof(float), "K.dat");
            //export_cuda_buffer(d_KCacheRemapIdx, num_vec, 1, sizeof(int), "KRemapIdx.dat");
            //export_cuda_buffer(d_KCacheRowIdx, cache_rows, 1, sizeof(int), "KRowIdx.dat");
            //int ws[WS];
            //assert_cuda(hipMemcpy(ws, d_workingset, WS * sizeof(int), hipMemcpyDeviceToHost));
            //std::sort(ws, ws + WS);
            //std::cout << "WS:";
            //for (int i = 0; i < WS; i++)
            //    std::cout << " " << ws[i];
            //std::cout << std::endl;
            //std::ostringstream ss;
            //ss << iter;
            //std::string siter = ss.str();
            //export_cuda_buffer(d_workingset, WS, 1, sizeof(int), "ws" + siter + ".dat");
            //export_cuda_buffer(d_y, num_vec, 1, sizeof(float), "y" + siter + ".dat");
            //export_cuda_buffer(d_g, num_vec, 1, sizeof(float), "g" + siter + ".dat");
            //export_cuda_buffer(d_alpha, num_vec, 1, sizeof(float), "alpha" + siter + ".dat");
            //export_cuda_buffer(d_workingset, WS, 1, sizeof(int), "ws.dat");

#ifdef USE_DAIFLETCHER
            dim3 dimBlockKtoH(32, 8);
            dim3 dimGridKtoH(getgriddim(WS, dimBlockKtoH.x), getgriddim(WS, dimBlockKtoH.y));
            kernelKtoHLocal<WS><<<dimGridKtoH, dimBlockKtoH>>>(d_KLocal, d_y, d_workingset);
            kernelCalcE<WS><<<1, WS>>>(d_workingset, d_alpha, d_y, num_vec);
            //float e;
            //assert_cuda(hipMemcpyFromSymbol(&e, HIP_SYMBOL(d_df_e), sizeof(e)));
            //std::cout << "===\ne: " << e << std::endl;
            //std::cout << "===\nalpha: ";
            //print_cuda_buffer(d_alpha, WS, 1, std::cout);
            //std::cout << "===\ng: ";
            //print_cuda_buffer(d_g, WS, 1, std::cout);
            //std::cout << "===\ny: ";
            //print_cuda_buffer(d_y, WS, 1, std::cout);
            kernelUpdateGHLocal<WS><<<1, WS>>>(d_df_gh, d_g, d_y, d_alpha, d_workingset, d_KLocal);
            //std::cout << "===\ngh: ";
            //print_cuda_buffer(d_df_gh, WS, 1, std::cout);
            //std::cout << "===\K row 0: ";
            //print_cuda_buffer(d_KLocal, WS, 1, std::cout);
            float DELTAvpm = 1e-3,
                DELTAkin = 1;
            //ACCUMULATE_KERNEL_TIME(timer_local_solver, counter_local_solver, (kernelFletcherAlg2A<WS><<<1, WS>>>(WS, d_KLocal, d_df_gh, C, d_y, d_alpha, d_alphadiff, DELTAvpm*DELTAkin, d_df_g, d_df_y, d_df_tempv, d_df_d, d_df_Ad, d_df_t, d_df_xplus, d_df_tplus, d_df_sk, d_df_yk)));
            ACCUMULATE_KERNEL_TIME(timer_local_solver, counter_local_solver, (kernelFletcherAlg2A<WS><<<1, WS>>>(d_KLocal, d_df_gh, C, d_y, d_alpha, d_alphadiff, DELTAvpm*DELTAkin, d_workingset)));
#else
#if 0
            kernelSMO1Block<true, true, false><<<1, dimBlock32x8>>>(d_x, d_x2, d_y, d_g, d_alpha, d_workingset, gamma, C, eps, num_vec, num_vec_aligned, dim, dim_aligned, d_K, d_KCacheRemapIdx);
#else
            if (ELEM_PER_THREAD > 1)
                ACCUMULATE_KERNEL_TIME(timer_local_solver, counter_local_solver, (kernelSMO1BlockV2N<WS, ELEM_PER_THREAD> << <1, WS / ELEM_PER_THREAD >> >(d_y, d_g, d_alpha, d_alphadiff, d_workingset, gamma, C, eps, num_vec_aligned, d_KLocal, d_KCacheRemapIdx)));
            else
                ACCUMULATE_KERNEL_TIME(timer_local_solver, counter_local_solver, (kernelSMO1BlockV2<WS> << <1, WS >> >(d_y, d_g, d_alpha, d_alphadiff, d_workingset, gamma, C, eps, num_vec_aligned, d_KLocal, d_KCacheRemapIdx)));
#endif
            //print_cuda_buffer(d_g, 10, 1, std::cout);
#endif
            //std::cout << "===\nalpha: ";
            //print_cuda_buffer(d_alpha, WS, 1, std::cout);
            //std::cout << "===\nAdiff: ";
            //print_cuda_buffer(d_alphadiff, WS, 1, std::cout);
#ifdef CALC_KLOCAL
            if (sparse)
            {
                ACCUMULATE_KERNEL_TIME(timer_check_cache, counter_check_cache, (checkCacheKLocal<findCacheRowBlockSize, WS, ELEM_PER_THREAD>(sparse, d_workingset, d_x, d_x2, sparse_data_gpu, ellpack_data_gpu, jds_data_gpu, use_ellpack, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_alphadiff, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma)));
            }
            else
            {
#ifdef USE_CUBLAS
                ACCUMULATE_KERNEL_TIME(timer_check_cache, counter_check_cache, (checkCacheCublasKLocal<findCacheRowBlockSize, WS, ELEM_PER_THREAD>(sparse, d_workingset, d_x, d_xT, d_xTile, d_x2, sparse_data_gpu, d_K, d_KTile, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_alphadiff, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma, cublas)));
#else
                ACCUMULATE_KERNEL_TIME(timer_check_cache, counter_check_cache, (checkCacheKLocal<findCacheRowBlockSize, WS, ELEM_PER_THREAD>(sparse, d_workingset, d_x, d_x2, sparse_data_gpu, ellpack_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_alphadiff, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma)));
#endif
            }
#endif
            ACCUMULATE_KERNEL_TIME(timer_Gupdate, counter_Gupdate, (kernelUpdateG<WS> << <dimGrid, dimBlock >> >(d_y, d_g, d_alphadiff, d_workingset, d_K, d_KCacheRemapIdx, num_vec, num_vec_aligned)));

            //export_cuda_buffer(d_alpha, num_vec, 1, sizeof(float), "alpha.dat");
            //std::cout << "===\ng: ";
            //print_cuda_buffer(d_g, WS, 1, std::cout);

            if ((iter + 1) % 100 == 0)
            {
                float diff;
                assert_cuda(hipMemcpyFromSymbol(&diff, HIP_SYMBOL(d_diff), sizeof(float), 0));
                //if (diff == last_diff)
                //{
                //    int ws[WS];
                //    assert_cuda(hipMemcpy(ws, d_workingset, WS * sizeof(int), hipMemcpyDeviceToHost));
                //    std::sort(ws, ws + WS);
                //    std::cout << "WS:";
                //    for (int i = 0; i < WS; i++)
                //        std::cout << " " << ws[i];
                //    std::cout << std::endl;
                //    break;
                //}
                //last_diff = diff;

                size_t block_num_iter;
                assert_cuda(hipMemcpyFromSymbol(&block_num_iter, HIP_SYMBOL(d_block_num_iter), sizeof(block_num_iter), 0));
                assert_cuda(hipMemcpyFromSymbol(&total_num_iter, HIP_SYMBOL(d_total_num_iter), sizeof(total_num_iter), 0));
                std::cout << "Iter: " << total_num_iter << ", global iter: " << iter << ", diff: " << diff << std::endl;
                //std::cout << "Local iter: " << block_num_iter << std::endl;

                if (block_num_iter >= MAX_BLOCK_ITER)
                    std::cout << "Warning: Maximum number of iterations per block was reached" << std::endl;

                if (block_num_iter == 0)
                {
                    assert_cuda(hipMemcpyFromSymbol(rho, HIP_SYMBOL(d_rho), sizeof(float), 0));
                    std::cout << "Optimality reached after " << iter << " iterations, stopping loop. rho = " << *rho << std::endl;
                    break;
                }
            }
            if (iter >= MAX_GLOBAL_ITERS)
            {
                //kernelFindActiveSet<findActiveSetBlocksize><<<dimGridFindActiveSet, dimBlockFindActiveSet>>>(d_workingset, d_y, d_g, d_alpha, C, num_vec_shrunk);
                //kernelFindCacheRowV2<256><<<1, 256>>>(d_workingset, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows);
                //int cache_rows_to_compute[WS];
                //int num_cache_rows_to_compute;
                //assert_cuda(hipMemcpyFromSymbol(&num_cache_rows_to_compute, HIP_SYMBOL(d_num_cache_rows_to_compute), sizeof(int), 0));
                //assert_cuda(hipMemcpyFromSymbol(cache_rows_to_compute, HIP_SYMBOL(d_cache_rows_to_compute), num_cache_rows_to_compute * sizeof(int), 0));
                //std::cout << "Num cache rows to compute: " << num_cache_rows_to_compute << "\n";
                //for (int i = 0; i < num_cache_rows_to_compute; i++)
                //    std::cout << " " << cache_rows_to_compute[i];
                //std::cout << std::endl;

                //std::cout << "WS: "; print_cuda_buffer(d_workingset, 10, 1);
                //std::cout << "K row 0: "; print_cuda_buffer(d_K, 10, 1);
                //std::cout << "K row 1: "; print_cuda_buffer(d_K + num_vec_aligned, 10, 1);
                //std::cout << "KCacheRowIdx: "; print_cuda_buffer(d_KCacheRowIdx, cache_rows, 1);

                //break; //LLLLLLLLLLLLLLLLLLLLLL
            }
        }

        assert_cuda(hipMemcpyFromSymbol(&cacheUpdateCnt, HIP_SYMBOL(d_cacheUpdateCnt), sizeof(int), 0));
        std::cout << "Cache row updates: " << cacheUpdateCnt << std::endl;
        assert_cuda(hipMemcpyFromSymbol(updateGCnt, HIP_SYMBOL(d_updateGCnt), sizeof(updateGCnt)));
        std::cout << "Update G: updated " << updateGCnt[0] << ", skipped " << updateGCnt[1] << std::endl;

        PRINT_KERNEL_TIME("CheckCache       ", timer_check_cache, counter_check_cache);
        PRINT_KERNEL_TIME("Find Active set  ", timer_find_active_set, counter_find_active_set);
        PRINT_KERNEL_TIME("Find N-best      ", timer_find_nbest, counter_find_nbest);
        PRINT_KERNEL_TIME("Fill working set ", timer_fill_ws, counter_fill_ws);
        PRINT_KERNEL_TIME("Calc/Copy KLocal ", timer_calc_klocal, counter_calc_klocal);
        PRINT_KERNEL_TIME("Local Solver     ", timer_local_solver, counter_local_solver);
        PRINT_KERNEL_TIME("G-update         ", timer_Gupdate, counter_Gupdate);
#ifdef USE_TIMERS
        std::cout << "Total device time       : " << (timer_check_cache
            + timer_find_active_set
            + timer_find_nbest
            + timer_fill_ws
            + timer_calc_klocal
            + timer_local_solver
            + timer_Gupdate) << " ms\n";
#endif

        assert_cuda(hipMemcpy(alpha, d_alpha, num_vec * sizeof(float), hipMemcpyDeviceToHost));

        if (sparse)
        {
            if (use_ellpack)
                freeCudaEllpack(ellpack_data_gpu);
            else
                freeCudaCsr(sparse_data_gpu);
            freeCudaJds(jds_data_gpu);
            assert_cuda(hipFree(d_denseVec));
        }
        else
        {
            assert_cuda(hipFree(d_x));
        }

        assert_cuda(hipFree(d_x2));
        assert_cuda(hipFree(d_K));
        assert_cuda(hipFree(d_KLocal));
        assert_cuda(hipFree(d_KDiag));
        assert_cuda(hipFree(d_KCacheRemapIdx));
        assert_cuda(hipFree(d_KCacheRowIdx));
        assert_cuda(hipFree(d_KCacheRowPriority));
        assert_cuda(hipFree(d_alpha));
        assert_cuda(hipFree(d_alphadiff));
        assert_cuda(hipFree(d_y));
        assert_cuda(hipFree(d_g));
        assert_cuda(hipFree(d_sortval));
        assert_cuda(hipFree(d_sortidx));
	    assert_cuda(hipFree(d_sortValues));
	    assert_cuda(hipFree(d_sortIdxs));
        assert_cuda(hipFree(d_workingset));
        assert_cuda(hipFree(d_ws_priority));
#ifdef USE_CUBLAS
        assert_cuda(hipFree(d_xTile));
        assert_cuda(hipFree(d_KTile));
        assert_cuda(hipFree(d_xT));
#endif
#ifdef USE_DAIFLETCHER
        assert_cuda(hipFree(d_df_g));
        assert_cuda(hipFree(d_df_gh));
        assert_cuda(hipFree(d_df_y));
        assert_cuda(hipFree(d_df_tempv));
        assert_cuda(hipFree(d_df_d));
        assert_cuda(hipFree(d_df_Ad));
        assert_cuda(hipFree(d_df_t));
        assert_cuda(hipFree(d_df_xplus));
        assert_cuda(hipFree(d_df_tplus));
        assert_cuda(hipFree(d_df_sk));
        assert_cuda(hipFree(d_df_yk));
#endif
        assert_cublas(hipblasDestroy(cublas));
    }
    catch (...)
    {
        if (sparse)
        {
            if (use_ellpack)
                freeCudaEllpack(ellpack_data_gpu);
            else
                freeCudaCsr(sparse_data_gpu);
            freeCudaJds(jds_data_gpu);
            hipFree(d_denseVec);
        }
        else
        {
            hipFree(d_x);
        }

        hipFree(d_x2);
        hipFree(d_K);
        hipFree(d_KLocal);
        hipFree(d_KDiag);
        hipFree(d_KCacheRemapIdx);
        hipFree(d_KCacheRowIdx);
        hipFree(d_KCacheRowPriority);
        hipFree(d_alpha);
        hipFree(d_alphadiff);
        hipFree(d_y);
        hipFree(d_g);
        hipFree(d_sortval);
        hipFree(d_sortidx);
	    hipFree(d_sortValues);
	    hipFree(d_sortIdxs);
        hipFree(d_workingset);
        hipFree(d_ws_priority);
#ifdef USE_CUBLAS
        hipFree(d_xTile);
        hipFree(d_KTile);
        hipFree(d_xT);
#endif
#ifdef USE_DAIFLETCHER
        hipFree(d_df_g);
        hipFree(d_df_gh);
        hipFree(d_df_y);
        hipFree(d_df_tempv);
        hipFree(d_df_d);
        hipFree(d_df_Ad);
        hipFree(d_df_t);
        hipFree(d_df_xplus);
        hipFree(d_df_tplus);
        hipFree(d_df_sk);
        hipFree(d_df_yk);
#endif
        hipblasDestroy(cublas);
        throw;
    }

}

void OrcusSvm1B::Train(float * alpha, float * rho, bool sparse, const Data & x, const float * y, size_t num_vec, size_t num_vec_aligned, size_t dim, size_t dim_aligned, float C, float gamma, float eps, int ws_size)
{
    if (ws_size == 0)
    {
        if (num_vec >= 250000)
            ws_size = 2048;
        else
            ws_size = 1024;
/*#ifdef USE_ELLPACK
        if (sparse)
        {
            size_t free_mem, total_mem;
            assert_cuda(hipFree(nullptr));  //force CUDA init
            assert_cuda(hipMemGetInfo(&free_mem, &total_mem));

            std::vector<int> rowLen(x.sparse->numRows);
            std::adjacent_difference(x.sparse->rowOffsets + 1, x.sparse->rowOffsets + x.sparse->numRows + 1, rowLen.begin());
            int maxRowLen = *std::max_element(rowLen.begin(), rowLen.end());
            size_t ellpack_size = 2 * x.sparse->numRows * maxRowLen * sizeof(float);

            if (ellpack_size >= free_mem * 0.5)
                ws_size = 512;
        }
#endif*/
    }
    ws_size = std::max(64, std::min(2048, ws_size));
    try
    {
        switch (ws_size)
        {
        case   64: train<  64>(alpha, rho, sparse, x, y, num_vec, num_vec_aligned, dim, dim_aligned, C, gamma, eps); break;
        case  128: train< 128>(alpha, rho, sparse, x, y, num_vec, num_vec_aligned, dim, dim_aligned, C, gamma, eps); break;
        case  256: train< 256>(alpha, rho, sparse, x, y, num_vec, num_vec_aligned, dim, dim_aligned, C, gamma, eps); break;
        case  512: train< 512>(alpha, rho, sparse, x, y, num_vec, num_vec_aligned, dim, dim_aligned, C, gamma, eps); break;
        case 1024: train<1024>(alpha, rho, sparse, x, y, num_vec, num_vec_aligned, dim, dim_aligned, C, gamma, eps); break;
        case 2048: train<2048>(alpha, rho, sparse, x, y, num_vec, num_vec_aligned, dim, dim_aligned, C, gamma, eps); break;
        default:
            std::cerr << "Unsupported working set size\n";
        }
    }
    catch (std::exception & e)
    {
        std::cerr << "Exception thrown: " << e.what() << std::endl;
        if (strstr(e.what(), "out of memory"))
            std::cerr << "Try lowering working set size\n";
    }
}
