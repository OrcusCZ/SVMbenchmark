#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <ctime>
#include <algorithm>
#include <vector>
#include <cfloat>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cassert>
#include "OrcusSvm.h"
#include "../cudaerror.h"
#include "../debug.h"

#if __cplusplus <= 199711L
#define nullptr NULL
#endif

#define assert_cuda_dev(call) \
{ \
    hipError_t e = (call); \
    if (e != hipSuccess) { \
        const char * s = hipGetErrorString(e); \
        printf("%s:%d: %s\n", __FILE__, __LINE__, s); \
        assert(0); \
    } \
}

extern int g_cache_size;

struct csr {
	unsigned int nnz;
	unsigned int numRows;
	unsigned int numCols;
	float *values;
	unsigned int *colInd;
	unsigned int *rowOffsets;
};

struct csr_gpu {
	unsigned int nnz;
	unsigned int numRows;
	unsigned int numCols;
	float *values;
	unsigned int *colInd;
	unsigned int *rowOffsets;
    unsigned int *rowLen;
};

template<typename T>
__host__ __device__ T getgriddim(T totallen, T blockdim)
{
    return (totallen + blockdim - (T)1) / blockdim;
}

template<typename T>
T rounduptomult(T x, T m)
{
    return ((x + m - (T)1) / m) * m;
}

#define DENSE_TILE_SIZE 16

namespace OrcusSVMDP
{
//init this to 0
__device__ int d_shrunkSize;
//init this to 0
__device__ int d_nonshrunkSize;
__device__ float2 d_shrinkMaxF;
__device__ int d_alphaStatusChange[2];

__device__ int d_cacheUpdateCnt;
//contains changes to KCacheRemapIdx buffer, which should be written after kernelCheckCache ends
//each change to buffer is contained in int2 variable (x,y) such that
//KCacheRemapIdx[x] = y
//pair at index [2] is for KCacheRowPriority
__device__ int2 d_KCacheChanges[3];

__device__ int d_cacheRow;
}

using namespace OrcusSVMDP;

template<typename T>
__device__ void swap_dev(T & a, T & b)
{
    T tmp = a;
    a = b;
    b = tmp;
}

template<typename T>
__global__ static void kernelMemset(T * mem, T v, int n)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    while (k < n)
    {
        mem[k] = v;
        k += gridDim.x * blockDim.x;
    }
}

template<typename T>
static void memsetCuda(T * d_mem, T v, int n)
{
    dim3 dimBlock(256);
    dim3 dimGrid(std::min(2048, getgriddim<int>(n, dimBlock.x)));
    kernelMemset<T><<<dimGrid, dimBlock>>>(d_mem, v, n);
}

__global__ static void kernelCalcRowLen(unsigned int * rowLen, const unsigned int * rowOffsets, int numRows)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    if (k < numRows)
    {
        rowLen[k] = rowOffsets[k + 1] - rowOffsets[k];
    }
}

__global__ static void kernelPow2(float * x2, const float * x, int w, int h, int pitch)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x,
        j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < w && j < h)
    {
        int idx = pitch * j + i;
        float v = x[idx];
        x2[idx] = v * v;
    }
}

__global__ static void kernelPow2SumSparse(csr_gpu x, float * x2)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    while (k < x.numRows)
    {
        float sum = 0;
        int end = x.rowOffsets[k + 1];
        for (int i = x.rowOffsets[k]; i < end; i++)
        {
            float v = x.values[i];
            sum += v * v;
        }
        x2[k] = sum;

        k += gridDim.x * blockDim.x;
    }
}

static void computeX2Dense(const float * d_x, float * d_x2sum, int num_vec, int num_vec_aligned, int dim, int dim_aligned, hipblasHandle_t cublas)
{
    float *d_x2 = nullptr,
        *d_ones = nullptr;
    assert_cuda(hipMalloc(&d_x2, num_vec_aligned * dim_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_ones, dim_aligned * sizeof(float)));
    memsetCuda<float>(d_ones, 1, dim_aligned);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(getgriddim(dim, (int)dimBlock.x), getgriddim(num_vec, (int)dimBlock.y));
    kernelPow2<<<dimGrid, dimBlock>>>(d_x2, d_x, dim, num_vec, dim_aligned);
    float a = 1,
        b = 0;
    assert_cublas(hipblasSgemv(cublas, HIPBLAS_OP_T, dim, num_vec, &a, d_x2, dim_aligned, d_ones, 1, &b, d_x2sum, 1));

    assert_cuda(hipFree(d_x2));
}

static void computeX2Sparse(csr_gpu & x, float * d_x2)
{
    dim3 dimBlock(256);
    dim3 dimGrid(std::min(256, getgriddim<int>(x.numRows, dimBlock.x)));
    kernelPow2SumSparse<<<dimGrid, dimBlock>>>(x, d_x2);
}

static void computeKDiag(float * d_KDiag, int num_vec)
{
    //K[i,i] is always 1 for RBF kernel, let's just use memset here
    memsetCuda<float>(d_KDiag, 1, num_vec);
}

__global__ static void kernelSelectI(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    extern __shared__ float sval[];
    int * sidx = (int *)(sval + blockDim.x);

    float max_val = -FLT_MAX;
    int max_idx = 0;

    for (int k = blockDim.x * blockIdx.x + threadIdx.x; k < num_vec; k += gridDim.x * blockDim.x)
    {
        float v;
        float y_ = y[k];
        float a_ = alpha[k];
        if ((y_ == 1 && a_ < C) || (y_ == -1 && a_ > 0))
            v = y[k] * g[k];
        else
            v = -FLT_MAX;
        if (v > max_val)
        {
            max_val = v;
            max_idx = k;
        }
    }

    sval[threadIdx.x] = max_val;
    sidx[threadIdx.x] = max_idx;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (sval[threadIdx.x + s] > sval[threadIdx.x])
            {
                sval[threadIdx.x] = sval[threadIdx.x + s];
                sidx[threadIdx.x] = sidx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        valbuf[blockIdx.x] = sval[0];
        idxbuf[blockIdx.x] = sidx[0];
    }
}

//first order search
__global__ static void kernelSelectJ1(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec)
{
    extern __shared__ float sval[];
    int * sidx = (int *)(sval + blockDim.x);;

    float max_val = -FLT_MAX;
    int max_idx = 0;

    for (int k = blockDim.x * blockIdx.x + threadIdx.x; k < num_vec; k += gridDim.x * blockDim.x)
    {
        float v;
        float y_ = y[k];
        float a_ = alpha[k];
        if ((y_ == 1 && a_ > 0) || (y_ == -1 && a_ < C))
            v = -y[k] * g[k]; //return negative, so we can use reducemax
        else
            v = -FLT_MAX;
        if (v > max_val)
        {
            max_val = v;
            max_idx = k;
        }
    }

    sval[threadIdx.x] = max_val;
    sidx[threadIdx.x] = max_idx;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (sval[threadIdx.x + s] > sval[threadIdx.x])
            {
                sval[threadIdx.x] = sval[threadIdx.x + s];
                sidx[threadIdx.x] = sidx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        valbuf[blockIdx.x] = sval[0];
        idxbuf[blockIdx.x] = sidx[0];
    }
}

//second order search with cached K
__global__ static void kernelSelectJCached(float * valbuf, int * idxbuf, const float * y, const float * g, const float * alpha, float C, int num_vec, int num_vec_aligned, const int * i_ptr, const float * K, const float * KDiag, const int * KCacheRemapIdx)
{
    extern __shared__ float sval[];
    int * sidx = (int *)(sval + blockDim.x);

    float max_val = -FLT_MAX;
    int max_idx = 0;

    for (int k = blockDim.x * blockIdx.x + threadIdx.x; k < num_vec; k += gridDim.x * blockDim.x)
    {
        int i = *i_ptr;
        int cache_row = KCacheRemapIdx[i];
        float val;
        float y_ = y[k];
        float a_ = alpha[k];
        float th = y[i] * g[i];
        if (((y_ == 1 && a_ > 0) || (y_ == -1 && a_ < C)) && th > y[k] * g[k])
        {
            float den = KDiag[i] + KDiag[k] - 2 * K[(size_t)num_vec_aligned * cache_row + k];
            float v = th - y[k] * g[k];
            val = v * v / den;
        }
        else
            val = -FLT_MAX;
        if (val > max_val)
        {
            max_val = val;
            max_idx = k;
        };
    }

    sval[threadIdx.x] = max_val;
    sidx[threadIdx.x] = max_idx;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (sval[threadIdx.x + s] > sval[threadIdx.x])
            {
                sval[threadIdx.x] = sval[threadIdx.x + s];
                sidx[threadIdx.x] = sidx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        valbuf[blockIdx.x] = sval[0];
        idxbuf[blockIdx.x] = sidx[0];
    }
}

//assume grid size (1)
__global__ static void kernelReduceMaxIdx(float * val, int * idx, int * idx_out, int len)
{
    extern __shared__ float sval[];
    int * sidx = (int *)(sval + blockDim.x);

    float max_val = -FLT_MAX;
    int max_idx = 0;

    for (int i = threadIdx.x; i < len; i += blockDim.x)
    {
        float v = val[i];
        if (v > max_val)
        {
            max_val = v;
            max_idx = idx[i];
        }
    }

    sval[threadIdx.x] = max_val;
    sidx[threadIdx.x] = max_idx;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (sval[threadIdx.x + s] > sval[threadIdx.x])
            {
                sval[threadIdx.x] = sval[threadIdx.x + s];
                sidx[threadIdx.x] = sidx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        idx_out[0] = sidx[0];
}

__device__ __host__ static void reduceMaxIdx(float * d_val, int * d_idx, int * d_result, int len, int reduce_block_size)
{
    //dim3 dimBlock(reduce_block_size);
    //dim3 dimGrid(std::min(reduce_block_size, getgriddim(len, (int)dimBlock.x)));
    //kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val, d_idx, d_val2, d_idx2, len);
    //len = dimGrid.x;
    //dimGrid.x = 1;
    //kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val2, d_idx2, d_val, d_idx, len);
    dim3 dimBlock(reduce_block_size);
    dim3 dimGrid(1);
    kernelReduceMaxIdx<<<dimGrid, dimBlock, dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int)>>>(d_val, d_idx, d_result, getgriddim(len, (int)dimBlock.x));
}

__global__ static void kernelUpdateg(float * g, const float * lambda, const float * y, const float * K, const int * ws, int num_vec, int num_vec_aligned)
{
    int i = ws[0];
    int j = ws[1];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        g[k] += *lambda * y[k] * (K[(size_t)num_vec_aligned * j + k] - K[(size_t)num_vec_aligned * i + k]);
    }
}

__global__ static void kernelUpdategCached(float * g, const float * lambda, const float * y, const float * K, const int * ws, int num_vec, int num_vec_aligned, const int * KCacheRemapIdx)
{
    int i = ws[0];
    int j = ws[1];
    int i_cache_row = KCacheRemapIdx[i];
    int j_cache_row = KCacheRemapIdx[j];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < num_vec)
    {
        g[k] += *lambda * y[k] * (K[(size_t)num_vec_aligned * j_cache_row + k] - K[(size_t)num_vec_aligned * i_cache_row + k]);
    }
}

__global__ static void kernelUpdateAlphaAndLambda(float * alpha, float * lambda, const float * y, const float * g, const float * K, float C, const int * ws, int num_vec, int num_vec_aligned)
{
    int i = ws[0];
    int j = ws[1];
    float l1 = y[i] > 0 ? C - alpha[i] : alpha[i];
    float l2 = y[j] > 0 ? alpha[j] : C - alpha[j];
    float l3 = (y[i] * g[i] - y[j] * g[j]) / (K[(size_t)num_vec_aligned * i + i] + K[(size_t)num_vec_aligned * j + j] - 2 * K[(size_t)num_vec_aligned * i + j]);
    float l = min(l1, min(l2, l3));

    *lambda = l;
    alpha[i] += l * y[i];
    alpha[j] -= l * y[j];
}

__global__ static void kernelUpdateAlphaAndLambdaCached(float * alpha, float * lambda, const float * y, const float * g, const float * K, float C, const int * ws, int num_vec_aligned, const float * KDiag, const int * KCacheRemapIdx)
{
    int i = ws[0];
    int j = ws[1];
    int cache_row = KCacheRemapIdx[i];
    float l1 = y[i] > 0 ? C - alpha[i] : alpha[i];
    float l2 = y[j] > 0 ? alpha[j] : C - alpha[j];
    float l3 = (y[i] * g[i] - y[j] * g[j]) / (KDiag[i] + KDiag[j] - 2 * K[(size_t)num_vec_aligned * cache_row + j]);
    float l = min(l1, min(l2, l3));

    *lambda = l;
    alpha[i] += l * y[i];
    alpha[j] -= l * y[j];
}

__global__ static void kernelCheckCacheFinalize(int * KCacheRemapIdx, int * KCacheRowPriority)
{
    int2 c;
#pragma unroll
    for (int i = 0; i < 2; i++)
    {
        c = d_KCacheChanges[i];
        if (c.x >= 0)
        {
            KCacheRemapIdx[c.x] = c.y;
            d_KCacheChanges[i].x = -1;
        }
    }
    c = d_KCacheChanges[2];
    if (c.x >= 0)
    {
        KCacheRowPriority[c.x] = c.y;
        d_KCacheChanges[2].x = -1;
    }
}

__global__ static void kernelCheckCachePriority(const int * i_ptr, float * K, int * KCacheRemapIdx, int * KCacheRowIdx, int * KCacheRowPriority, int cache_rows, const float * x, const float * xT, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int last = d_cacheRow;
    if (last < 0)
        return;
    extern __shared__ int2 spriority[];
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = *i_ptr;

    //calculate cache matrix row [last], original index is [i]
    float * sx = (float *)spriority;
    for (int idxshift = 0; idxshift < dim; idxshift += blockDim.x)
    {
        int idx = idxshift + threadIdx.x;
        if (idx < dim)
            sx[idx] = x[dim_aligned * i + idx];
    }
    __syncthreads();
    while (j < num_vec)
    {
        float sum = 0;
        for (int d = 0; d < dim; d++)
        {
            float diff = sx[d] - xT[num_vec_aligned * d + j];
            sum += diff * diff;
        }
        K[(size_t)num_vec_aligned * last + j] = expf(-gamma * sum);
        j += gridDim.x * blockDim.x;
    }
}

__global__ static void kernelCheckCachePriorityV2(const int * i_ptr, float * K, int * KCacheRemapIdx, int * KCacheRowIdx, int * KCacheRowPriority, int cache_rows, const float * x, const float * x2, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int last = d_cacheRow;
    if (last < 0)
        return;
    __shared__ float shsum[DENSE_TILE_SIZE][DENSE_TILE_SIZE+1];
    int block = blockDim.x * blockIdx.x;
    int i = *i_ptr;

    //calculate cache matrix row [last], original index is [i]
    while (block < num_vec)
    {
        int j = block + threadIdx.y;
        int jout = block + threadIdx.x;
        float sum = 0;
        if (j < num_vec)
        {
            for (int d = threadIdx.x; d < dim; d += DENSE_TILE_SIZE)
            {
                sum += x[dim_aligned * i + d] * x[dim_aligned * j + d];
            }
        }
        shsum[threadIdx.y][threadIdx.x] = sum;
        __syncthreads();
        for (int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (threadIdx.x < s)
                shsum[threadIdx.y][threadIdx.x] += shsum[threadIdx.y][threadIdx.x + s];
            __syncthreads();
        }
        if (threadIdx.y == 0 && jout < num_vec)
        {
            sum = x2[i] + x2[jout] - 2 * shsum[threadIdx.x][0];
            K[(size_t)num_vec_aligned * last + jout] = expf(-gamma * sum);
        }
        __syncthreads();
        block += gridDim.x * blockDim.x;
    }
}

__global__ static void kernelMakeDenseVec(const int * i_ptr, const int * KCacheRemapIdx, csr_gpu x, float * vec)
{
    int i = *i_ptr;
    if (KCacheRemapIdx[i] >= 0)  //if [i] is already in cache, exit. we won't need any dense vector
        return;
    int j = x.rowOffsets[i] + blockDim.x * blockIdx.x + threadIdx.x;
    while (j < x.rowOffsets[i + 1])
    {
        vec[x.colInd[j]] = x.values[j];
        j += gridDim.x * blockDim.x;
    }
}

__global__ static void kernelFindCacheRow(const int * i_ptr, int * KCacheRemapIdx, int * KCacheRowIdx, int * KCacheRowPriority, int cache_rows)
{
    extern __shared__ int2 spriority[];
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = *i_ptr;
    if (KCacheRemapIdx[i] >= 0)
    {
        if (j == 0)
        {
            KCacheRowPriority[KCacheRemapIdx[i]] = d_cacheUpdateCnt;  // refresh priority
            d_cacheRow = -1;
        }
        return;  //item already in cache
    }
    int2 minpriority = make_int2(INT_MAX, 0);
    for (int k = 0; k < cache_rows; k += blockDim.x)
    {
        int idx = k + threadIdx.x;
        if (idx < cache_rows)
        {
            int v = KCacheRowPriority[idx];
            if (v < minpriority.x)
                minpriority = make_int2(v, idx);
        }
    }
    spriority[threadIdx.x] = minpriority;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (spriority[threadIdx.x + s].x < spriority[threadIdx.x].x)
                spriority[threadIdx.x] = spriority[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (j == 0)
    {
        int last = spriority[0].y;
        int del_i = KCacheRowIdx[last];
        if (del_i >= 0)
            d_KCacheChanges[1] = make_int2(del_i, -1);  //cache row for vector [del_i] will be overwritten, remove it from RemapIdx array
        //set correct indices
        d_KCacheChanges[0] = make_int2(i, last);
        KCacheRowIdx[last] = i;
        d_KCacheChanges[2] = make_int2(last, ++d_cacheUpdateCnt);
        d_cacheRow = last;
    }
}

__global__ static void kernelCheckCacheSparsePriority(const int * i_ptr, float * K, int * KCacheRemapIdx, int * KCacheRowIdx, int * KCacheRowPriority, int cache_rows, const float * vec, const float * x2, csr_gpu x, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int last = d_cacheRow;
    if (last < 0)
        return;
    extern __shared__ int2 spriority[];
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = *i_ptr;

    //calculate cache matrix row [last], original index is [i]
    while (j < num_vec)
    {
        float sum = 0;
        int end = x.rowOffsets[j + 1];
        for (int d = x.rowOffsets[j]; d < end; d++)
        {
            sum += vec[x.colInd[d]] * x.values[d];
        }
        sum = x2[i] + x2[j] - 2 * sum;

        K[(size_t)num_vec_aligned * last + j] = expf(-gamma * sum);
        j += gridDim.x * blockDim.x;
    }
}

#define SPARSE_TILE_SIZE 16

__global__ static void kernelCheckCacheSparsePriorityV2(const int * i_ptr, float * K, int * KCacheRemapIdx, int * KCacheRowIdx, int * KCacheRowPriority, int cache_rows, const float * vec, const float * x2, csr_gpu x, float gamma, int num_vec, int num_vec_aligned, int dim, int dim_aligned)
{
    int last = d_cacheRow;
    if (last < 0)
        return;
    __shared__ float shsum[SPARSE_TILE_SIZE][SPARSE_TILE_SIZE+1];
    int block = blockDim.x * blockIdx.x;
    int i = *i_ptr;

    //calculate cache matrix row [last], original index is [i]
    while (block < num_vec)
    {
        int j = block + threadIdx.y;
        int jout = block + threadIdx.x;
        float sum = 0;
        if (j < num_vec)
        {
            //int end = x.rowOffsets[j + 1];
            int end = x.rowOffsets[j] + x.rowLen[j];
            for (int d = x.rowOffsets[j] + threadIdx.x; d < end; d += SPARSE_TILE_SIZE)
            {
                sum += vec[x.colInd[d]] * x.values[d];
            }
        }
        shsum[threadIdx.y][threadIdx.x] = sum;
        __syncthreads();
        for (int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (threadIdx.x < s)
                shsum[threadIdx.y][threadIdx.x] += shsum[threadIdx.y][threadIdx.x + s];
            __syncthreads();
        }
        if (threadIdx.y == 0 && jout < num_vec)
        {
            sum = x2[i] + x2[jout] - 2 * shsum[threadIdx.x][0];
            K[(size_t)num_vec_aligned * last + jout] = expf(-gamma * sum);
        }
        __syncthreads();
        block += gridDim.x * blockDim.x;
    }
}

//make a GPU deep copy of a CPU csr matrix
static hipError_t make_gpu_csr(csr_gpu &x_gpu, const csr &x_cpu) {
	x_gpu.nnz = x_cpu.nnz;
	x_gpu.numCols = x_cpu.numCols;
	x_gpu.numRows = x_cpu.numRows;

	assert_cuda(hipMalloc((void **)&(x_gpu.values), x_gpu.nnz * sizeof(float)));
	assert_cuda(hipMalloc((void **)&(x_gpu.colInd), x_gpu.nnz * sizeof(int)));
	assert_cuda(hipMalloc((void **)&(x_gpu.rowOffsets), (x_gpu.numRows+1) * sizeof(int)));
    assert_cuda(hipMalloc((void **)&(x_gpu.rowLen), x_gpu.numRows * sizeof(int)));

	assert_cuda(hipMemcpy(x_gpu.values, x_cpu.values, x_gpu.nnz * sizeof(float), hipMemcpyHostToDevice));
	assert_cuda(hipMemcpy(x_gpu.colInd, x_cpu.colInd, x_gpu.nnz * sizeof(int), hipMemcpyHostToDevice));
	assert_cuda(hipMemcpy(x_gpu.rowOffsets, x_cpu.rowOffsets, (x_gpu.numRows+1) * sizeof(int), hipMemcpyHostToDevice));

    dim3 dimBlock(256);
    dim3 dimGrid(getgriddim(x_gpu.numRows, dimBlock.x));
    kernelCalcRowLen<<<dimGrid, dimBlock>>>(x_gpu.rowLen, x_gpu.rowOffsets, x_gpu.numRows);

	return hipSuccess;
} //make_gpu_csr

static hipError_t cudaCsrFree(csr_gpu &x_gpu) {
	assert_cuda(hipFree(x_gpu.values));
	assert_cuda(hipFree(x_gpu.colInd));
	assert_cuda(hipFree(x_gpu.rowOffsets));
    assert_cuda(hipFree(x_gpu.rowLen));
	x_gpu.values = NULL;
	x_gpu.colInd = NULL;
	x_gpu.rowOffsets = NULL;
	x_gpu.nnz = 0;
	x_gpu.numRows = 0;
	x_gpu.numCols = 0;

	return hipSuccess;
} //cudaCsrFree

__host__ __device__ void checkCache(bool sparse, int * d_i, float * d_x, const float * d_x2, const csr_gpu & sparse_data_gpu, float * d_K, int * d_KCacheRemapIdx, int * d_KCacheRowIdx, int * d_KCacheRowPriority, float * d_denseVec, int num_vec, int num_vec_aligned, int dim, int dim_aligned, int cache_rows, float gamma)
{
    dim3 dimBlockCache(256);
    dim3 dimGridCache(getgriddim<int>(num_vec, dimBlockCache.x));
    size_t kernelCheckCacheSMSize = dimBlockCache.x * sizeof(int2);
    if (sparse)
    {
        assert_cuda_dev(hipMemsetAsync(d_denseVec, 0, dim * sizeof(float)));
        dim3 dimBlock(256);
        dim3 dimGrid(min(64, getgriddim<int>(dim, dimBlock.x)));
        kernelMakeDenseVec<<<dimGrid, dimBlock>>>(d_i, d_KCacheRemapIdx, sparse_data_gpu, d_denseVec);
        kernelFindCacheRow<<<1, 256, 256 * sizeof(int2)>>>(d_i, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows);
        dimBlock = dim3(SPARSE_TILE_SIZE, SPARSE_TILE_SIZE);
        dimGrid = dim3(min(256, getgriddim<int>(num_vec, dimBlock.y)));
        kernelCheckCacheSparsePriorityV2<<<dimGrid, dimBlock>>>(d_i, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows, d_denseVec, d_x2, sparse_data_gpu, gamma, num_vec, num_vec_aligned, dim, dim_aligned);
    }
    else
    {
        kernelCheckCacheSMSize = max(kernelCheckCacheSMSize, dim * sizeof(float));
        kernelFindCacheRow<<<1, 256, 256 * sizeof(int2)>>>(d_i, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows);
        dim3 dimBlock(DENSE_TILE_SIZE, DENSE_TILE_SIZE);
        dim3 dimGrid(min(256, getgriddim<int>(num_vec, dimBlock.y)));
        kernelCheckCachePriorityV2<<<dimGrid, dimBlock>>>(d_i, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, cache_rows, d_x, d_x2, gamma, num_vec, num_vec_aligned, dim, dim_aligned);
    }
    kernelCheckCacheFinalize<<<1, 1>>>(d_KCacheRemapIdx, d_KCacheRowPriority);
}

__global__ void kernelIterate(int num_iter, bool sparse, float * d_x, const float * d_x2, const float * d_y, float * d_alpha, float * d_g, float * d_lambda, csr_gpu sparse_data_gpu, float * d_K, const float * d_KDiag, int * d_KCacheRemapIdx, int * d_KCacheRowIdx, int * d_KCacheRowPriority, float * d_denseVec, int num_vec, int num_vec_aligned, int dim, int dim_aligned, int cache_rows, float gamma, float C, float * d_reduceval, int * d_reduceidx, int * d_workingset, int reduce_block_size)
{
    dim3 dimBlock(reduce_block_size);
    dim3 dimGrid(getgriddim<size_t>(num_vec, dimBlock.x));
    size_t sharedSizeSelect = dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int);
    for (int iter = 0; iter < num_iter; iter++)
    {
        kernelSelectI<<<dimGrid, dimBlock, sharedSizeSelect>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec);
        reduceMaxIdx(d_reduceval, d_reduceidx, d_workingset, num_vec, reduce_block_size);

        //check if I is cached
        checkCache(sparse, d_workingset, d_x, d_x2, sparse_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec, num_vec_aligned, dim, dim_aligned, cache_rows, gamma);

        kernelSelectJCached<<<dimGrid, dimBlock, sharedSizeSelect>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec, num_vec_aligned, d_workingset, d_K, d_KDiag, d_KCacheRemapIdx);
        reduceMaxIdx(d_reduceval, d_reduceidx, d_workingset + 1, num_vec, reduce_block_size);

        //check if J is cached
        checkCache(sparse, d_workingset + 1, d_x, d_x2, sparse_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec, num_vec_aligned, dim, dim_aligned, cache_rows, gamma);

        kernelUpdateAlphaAndLambdaCached<<<1, 1>>>(d_alpha, d_lambda, d_y, d_g, d_K, C, d_workingset, num_vec_aligned, d_KDiag, d_KCacheRemapIdx);
        kernelUpdategCached<<<dimGrid, dimBlock>>>(d_g, d_lambda, d_y, d_K, d_workingset, num_vec, num_vec_aligned, d_KCacheRemapIdx);
        assert_cuda_dev(hipDeviceSynchronize());
    }
}

void OrcusSvmTrain(float * alpha, float * rho, bool sparse, const float * x, const float * y, size_t num_vec, size_t num_vec_aligned, size_t dim, size_t dim_aligned, float C, float gamma, float eps)
{
    float *d_alpha = nullptr,
        *d_x = nullptr,
        *d_y = nullptr,
        *d_g = nullptr,
        *d_gBar = nullptr,
        *d_K = nullptr,
        *d_KDiag = nullptr,
        *d_reduceval = nullptr;
    int *d_reduceidx = nullptr;
    //TODO: move lambda and workingset to __device__ variables, no need for dynamic allocation
    float *d_lambda = nullptr;
    int *d_workingset = nullptr,
        *d_KCacheRemapIdx = nullptr,
        *d_KCacheRowIdx = nullptr,  // items at index [cache_rows] and [cache_rows+1] are indices of last inserted item
        *d_KCacheRowPriority = nullptr;  // the higher the priority is, the later was the item added
    float *d_denseVec = nullptr;  //dense vector used to calculate K cache row for sparse data
    float *d_x2 = nullptr;

    bool useShrinking = true;
    size_t reduce_block_size = 256;
    size_t reduce_buff_size = rounduptomult(num_vec, reduce_block_size);
    size_t ones_size = std::max(num_vec_aligned, dim_aligned);
    size_t cache_size_mb = g_cache_size;
    if (cache_size_mb == 0)
    {
        size_t free_mem, total_mem;
        assert_cuda(hipFree(nullptr));  //force CUDA init
        assert_cuda(hipMemGetInfo(&free_mem, &total_mem));
        cache_size_mb = free_mem / (1024 * 1024) - 200;  //leave 200 MB free
    }
    size_t cache_rows = cache_size_mb * 1024 * 1024 / (num_vec_aligned * sizeof(float));
    cache_rows = std::min(cache_rows, num_vec);

    std::cout << "Training data: " << (sparse ? "sparse" : "dense") << std::endl;
    std::cout << "Data size: " << num_vec << "\nDimension: " << dim << std::endl;
    std::cout << "Cache size: " << cache_rows << " rows (" << (100.f * cache_rows / (float)num_vec) << " % of data set)" << std::endl;

    hipblasHandle_t cublas;
    assert_cublas(hipblasCreate(&cublas));

    const csr * sparse_data = (const csr *)x;
    csr_gpu sparse_data_gpu;
    assert_cuda(hipMalloc(&d_x2, num_vec * sizeof(float)));
    if (sparse)
    {
        assert_cuda(make_gpu_csr(sparse_data_gpu, *sparse_data));
        assert_cuda(hipMalloc(&d_denseVec, dim * sizeof(float)));
        std::cout << "Precalculating X2" << std::endl;
        computeX2Sparse(sparse_data_gpu, d_x2);
    }
    else
    {
        assert_cuda(hipMalloc(&d_x, num_vec_aligned * dim_aligned * sizeof(float)));
    }
    assert_cuda(hipMalloc(&d_alpha, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_y, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_g, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_gBar, num_vec_aligned * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceval, reduce_buff_size / reduce_block_size * sizeof(float)));
    assert_cuda(hipMalloc(&d_reduceidx, reduce_buff_size / reduce_block_size * sizeof(int)));
    assert_cuda(hipMalloc(&d_lambda, sizeof(float)));
    assert_cuda(hipMalloc(&d_workingset, 2 * sizeof(int)));
    assert_cuda(hipMalloc(&d_KCacheRemapIdx, num_vec * sizeof(int)));
    assert_cuda(hipMalloc(&d_KCacheRowIdx, (cache_rows + 2) * sizeof(int)));  //last 2 items are indices of last cache row
    assert_cuda(hipMalloc(&d_KCacheRowPriority, cache_rows * sizeof(int)));
    assert_cuda(hipMalloc(&d_KDiag, num_vec * sizeof(float)));
    assert_cuda(hipMalloc(&d_K, cache_rows * num_vec_aligned * sizeof(float)));
    std::cout << "Cache size: " << (cache_rows * num_vec_aligned) << " floats\n";

    assert_cuda(hipMemset(d_alpha, 0, num_vec_aligned * sizeof(float)));
    if (!sparse)
    {
        assert_cuda(hipMemcpy(d_x, x, num_vec_aligned * dim_aligned * sizeof(float), hipMemcpyHostToDevice));
        std::cout << "Precalculating X2" << std::endl;
        computeX2Dense(d_x, d_x2, num_vec, num_vec_aligned, dim, dim_aligned, cublas);
    }
    assert_cuda(hipMemcpy(d_y, y, num_vec_aligned * sizeof(float), hipMemcpyHostToDevice));

    int KCacheChanges[6];
    for (int i = 0; i < sizeof(KCacheChanges) / sizeof(*KCacheChanges); i++)
        KCacheChanges[i] = -1;
    assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_KCacheChanges), KCacheChanges, sizeof(KCacheChanges), 0));

    memsetCuda<float>(d_g, 1, num_vec_aligned);
    assert_cuda(hipMemset(d_gBar, 0, num_vec_aligned * sizeof(float)));
    memsetCuda<int>(d_KCacheRemapIdx, -1, num_vec);
    memsetCuda<int>(d_KCacheRowIdx, -1, cache_rows + 2);
    memsetCuda<int>(d_KCacheRowPriority, -1, cache_rows);
    int cacheUpdateCnt = 0;
    assert_cuda(hipMemcpyToSymbol(HIP_SYMBOL(d_cacheUpdateCnt), &cacheUpdateCnt, sizeof(int), 0));

    std::cout << "Precalculating KDiag" << std::endl;
    computeKDiag(d_KDiag, num_vec);

    size_t num_vec_shrunk = num_vec;

    dim3 dimBlock(reduce_block_size);
    dim3 dimGrid(getgriddim(num_vec, (size_t)dimBlock.x));
    size_t sharedSizeSelect = dimBlock.x * sizeof(float) + dimBlock.x * sizeof(int);
    std::cout << "Starting iterations" << std::endl;
    int iter_step = 1000;
    for (int iter = 0;; iter += iter_step)
    {
        kernelIterate<<<1, 1>>>(iter_step, sparse, d_x, d_x2, d_y, d_alpha, d_g, d_lambda, sparse_data_gpu, d_K, d_KDiag, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma, C, d_reduceval, d_reduceidx, d_workingset, reduce_block_size);

        kernelSelectI<<<dimGrid, dimBlock, sharedSizeSelect>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec_shrunk);
        reduceMaxIdx(d_reduceval, d_reduceidx, d_workingset, num_vec_shrunk, reduce_block_size);

        //check if I is cached
        checkCache(sparse, d_workingset, d_x, d_x2, sparse_data_gpu, d_K, d_KCacheRemapIdx, d_KCacheRowIdx, d_KCacheRowPriority, d_denseVec, num_vec_shrunk, num_vec_aligned, dim, dim_aligned, cache_rows, gamma);

        kernelSelectJCached<<<dimGrid, dimBlock, sharedSizeSelect>>>(d_reduceval, d_reduceidx, d_y, d_g, d_alpha, C, num_vec_shrunk, num_vec_aligned, d_workingset, d_K, d_KDiag, d_KCacheRemapIdx);
        reduceMaxIdx(d_reduceval, d_reduceidx, d_workingset + 1, num_vec_shrunk, reduce_block_size);

        int ws[2];
        float yi, yj, gi, gj;
        assert_cuda(hipMemcpy(&ws, d_workingset, 2 * sizeof(int), hipMemcpyDeviceToHost));
        assert_cuda(hipMemcpy(&yi, d_y + ws[0], sizeof(float), hipMemcpyDeviceToHost));
        assert_cuda(hipMemcpy(&yj, d_y + ws[1], sizeof(float), hipMemcpyDeviceToHost));
        assert_cuda(hipMemcpy(&gi, d_g + ws[0], sizeof(float), hipMemcpyDeviceToHost));
        assert_cuda(hipMemcpy(&gj, d_g + ws[1], sizeof(float), hipMemcpyDeviceToHost));
        float diff = yi * gi - yj * gj;
        std::cout << "Iter " << iter << ": " << diff << " [" << ws[0] << "," << ws[1] << "]" << std::endl;
        if (diff < eps)
        {
            *rho = -(yi * gi + yj * gj) / 2;
            std::cout << "Optimality reached, stopping loop. rho = " << *rho << std::endl;
            break;
        }
    }

    assert_cuda(hipMemcpyFromSymbol(&cacheUpdateCnt, HIP_SYMBOL(d_cacheUpdateCnt), sizeof(int), 0));
    std::cout << "Cache row updates: " << cacheUpdateCnt << std::endl;

    assert_cuda(hipMemcpy(alpha, d_alpha, num_vec * sizeof(float), hipMemcpyDeviceToHost));

    if (sparse)
    {
        cudaCsrFree(sparse_data_gpu);
        assert_cuda(hipFree(d_denseVec));
    }
    else
    {
        assert_cuda(hipFree(d_x));
    }

    assert_cuda(hipFree(d_x2));
    assert_cuda(hipFree(d_K));
    assert_cuda(hipFree(d_KDiag));
    assert_cuda(hipFree(d_KCacheRemapIdx));
    assert_cuda(hipFree(d_KCacheRowIdx));
    assert_cuda(hipFree(d_KCacheRowPriority));
    assert_cuda(hipFree(d_alpha));
    assert_cuda(hipFree(d_y));
    assert_cuda(hipFree(d_g));
    assert_cuda(hipFree(d_gBar));
    assert_cuda(hipFree(d_reduceval));
    assert_cuda(hipFree(d_reduceidx));
    assert_cuda(hipFree(d_lambda));
    assert_cublas(hipblasDestroy(cublas));
}
